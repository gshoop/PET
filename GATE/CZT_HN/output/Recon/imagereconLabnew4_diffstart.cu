#include "hip/hip_runtime.h"

#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <queue>
#include <stdlib.h>
#include <math.h>
#include <cmath>
#include <list>
#include <cstdlib>
#include <sstream>
//#include <random>
#include <iomanip>
#include <limits>
#include <hip/hip_runtime.h>
#include "hip/device_functions.h"
#include <sys/time.h>
using namespace std;

#define sharesize 110
#define blocksPerGrid 30
#define threadsperBlock 32
#define ThreshLineValue 0.001
#define SRTWO 1.414
#define reducsize 1024
#define UpholdVox 20
#define CylRadius 55.0
#define CylHeight 126.0
#define MU 0.0096

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

class TimingCPU {

    private:
        long cur_time_;

    public:

        TimingCPU(): cur_time_(0) {};

        ~TimingCPU() {};

        void StartCounter(){
			struct timeval time;
			if(gettimeofday( &time, 0 )) return;
			cur_time_ = 1000000 * time.tv_sec + time.tv_usec;
		}

        double GetCounter(){
			struct timeval time;
			if(gettimeofday( &time, 0 )) return -1;

			long cur_time = 1000000 * time.tv_sec + time.tv_usec;
			double sec = (cur_time - cur_time_) / 1000000.0;
			if(sec < 0) sec += 86400;
		    cur_time_ = cur_time;
		
		    return 1000.*sec; //wall clock time (ms)
		}

} timing; // CPU timer.

struct PrivateTimingGPU {
    hipEvent_t     start;
    hipEvent_t     stop;
};

class TimingGPU {
    private:
        PrivateTimingGPU *privateTimingGPU;

    public:

        TimingGPU() { privateTimingGPU = new PrivateTimingGPU; }

        ~TimingGPU() {}

        void StartCounter() {
            hipEventCreate(&((*privateTimingGPU).start));
            hipEventCreate(&((*privateTimingGPU).stop));
            hipEventRecord((*privateTimingGPU).start,0);
        }

        float GetCounter(){
            float   time;
            hipEventRecord((*privateTimingGPU).stop, 0);
            hipEventSynchronize((*privateTimingGPU).stop);
            hipEventElapsedTime(&time,(*privateTimingGPU).start,(*privateTimingGPU).stop);
            return time;
        }

} timinggpu; // TimingGPU class

class timevar{
	public:
		float txforward;
		float txbackward;
		float tyforward;
		float tybackward;
		float tzforward;
		float tzbackward;
		float tpostimageprocess;
		float memoryIO;
		float lorsorting;
		timevar(): txforward(0.0), txbackward(0.0), tyforward(0.0), tybackward(0.0), tzforward(0.0), tzbackward(0.0), tpostimageprocess(0.0), memoryIO(0.0), lorsorting(0.0) {}
		~timevar() {}
	
		void printvalue(){
			printf("%12s%12s%12s%12s%12s%12s%20s%12s%12s", "(ms) xf","yf", "zf","xb", "yb","zb","postprocess","memoryIO","lorsorting\n");
			printf("%12.1f%12.1f%12.1f%12.1f%12.1f%12.1f%20.1f%12.1f%12.1f\n", txforward, tyforward, tzforward, txbackward, tybackward, tzbackward, tpostimageprocess, memoryIO, lorsorting);
	}
	
} timeall;

//__constant__ int *imageindex;
//__constant__ int *lorindex;
//__constant__ float *info;
__device__ __constant__ float aves[1] , avep[1], aveunf[1], d_bsgm[3], d_rads[3], d_info[4], d_norm[2];
__device__ __constant__ int d_indr[3], d_imageindex[4], d_lorindex[3];

struct lor
{
	float x1;
	float y1;
	float z1;
	float x2;
	float y2;
	float z2;
	int mainaxis;	//0 for x, 1 for y, 2 for z
	float weight;
};

struct cudalor
{
	float *x1;
	float *y1;
	float *z1;
	float *x2;
	float *y2;
	float *z2;
	float *linevalue;
	float *weight;
} xlor, ylor, zlor, dev_xlor, dev_ylor, dev_zlor;

int numcal=0, numcal2=0;

vector<string> explode(string s, char c);
int gaussianblur(int nx, int ny, int nz);
int preplor(string fin, int senmap);
__global__ void calnewmatrix000(float *snmatrix, float *smatrix);
__global__ void calnewmatrix100(float *snmatrix, float *smatrix, float *normimage);
__global__ void calnewmatrix010(float *snmatrix, float *smatrix, float *poimage);
__global__ void calnewmatrix110(float *snmatrix, float *smatrix, float *normimage, float *poimage);
__global__ void calnewmatrix011(float *snmatrix, float *smatrix, float *poimage, float *bmatrix, float *allweight);
__global__ void calnewmatrix111(float *snmatrix, float *smatrix, float *normimage, float *poimage, float *bmatrix, float *allweight);
__global__ void calave(float *smatrix, float *gave);
__global__ void calavewithfilter(float *smatrix, float *gave, int *gnumave);
__global__ void gpublur(float *smatrix, float *bmatrix, float *allweight);
__global__ void calLogLike(float *xlinevalue, double *gloglike, const int lorindex);	
__global__ void calLogLikeS(float *smatrix, float *normimage, double *gloglike, const int msize, const int norma);
__global__ void calLogR(float *smatrix, float *poimage, double *gloglike, const int msize);


float a = 4.; //grid size for sensitivity map.
float bndry[3] = {200., 200., 216.};    //FOV x, y, z. Unit: mm.
int msize;
float torhw, torsgm;
float beta, sumpop, sumimp, sumb;
int rgl;	//indicator for regularization. 0: no regularization, 1: regularization
int blur;	//indicator for blurring. 0: no blurring, 1: blurring in regularization
int norma; //indicator for normalization. 0: no normalization, 1: normalization using data and generate sensitivity image, 2: normalization using sensitivity image
float ThreshNorm;	//threshold for normalization
float bsgm[3], bthresh=0.01;	//sigma x,y,z in blurring Gaussian function
float deltax = 0.0001;  //delta used in calculating derivative numerically
float rads[3];	//radius for Gaussian blurring
int indr[3];	//number of grid for the radius

float *smatrix;  //matrix for image
float *snmatrix; //matrix for image
float *poimage; //prior image
float *lastimage; //prior image
float *bmatrix;	//blurred image of smatrix
float *deri;    //derivative of regularization function
float *allweight;	//all the sumed weight in blurred image.
float *normimage, *dev_normimage;	//normalization image (sensitivity image)

int numline = 0;    //number of lines/lors in each input file.
int *nummainaxis;   //number of lines in each main axis (x,y,z)
int wgt = 0;	//whether include attenuation and normalization in system matrix.
	

//__global__ void testf(float *dev_test)
//{
//	//if(threadIdx.x == 3) dev_test[0] = threadIdx.x + 1;
//	dev_test[0] = powf(maxs[0],2);
//}

__global__ void xfpro( cudalor lor, float *smatrix ) 
{
	int nx = d_imageindex[0], ny = d_imageindex[1], nz = d_imageindex[2];
	float a = d_info[0], torhw = d_info[1], torsgm2 = d_info[2];
	int lornum = d_lorindex[0];

	__shared__ float cache[sharesize][sharesize];
	int tid ;
	int cacheIndex1 , cacheIndex2;

	float linevalue = 0.;
	float ulen2, t, oy, oz;
	int mlyy,mhyy,mlzz,mhzz;
	float x1,x2,y1,y2,z1,z2,weight;

	int tilenum1 = (ny + sharesize - 1)/ sharesize, tilenum2 = (nz + sharesize - 1) / sharesize ;

	for(int i=0; i< nx; i++)
	{

        for(int tn1 = 0; tn1 < tilenum1; tn1++)	//for each tile of image. This is due to limited shared memory.
        {
		for(int tn2 = 0; tn2 < tilenum2; tn2++)
		{

        cacheIndex1 = threadIdx.x;

		while(cacheIndex1 < sharesize && ((sharesize * tn1) + cacheIndex1) < ny )	//load a tile of image
        {
		cacheIndex2 = threadIdx.y;
        while(cacheIndex2 < sharesize && ((sharesize * tn2) + cacheIndex2) < nz)
        {
            cache[cacheIndex1][cacheIndex2] = smatrix[i + ((sharesize * tn1) + cacheIndex1) * nx + ((sharesize * tn2) + cacheIndex2) * nx * ny];
            cacheIndex2 += blockDim.y;
        }
            cacheIndex1 += blockDim.x;
        }
        __syncthreads();

		tid = threadIdx.x + threadIdx.y * blockDim.x +  blockIdx.x * blockDim.x * blockDim.y;

		while(tid < lornum)
		{
			x1 = lor.x1[tid];
			y1 = lor.y1[tid];
			z1 = lor.z1[tid];
			x2 = lor.x2[tid];
			y2 = lor.y2[tid];
			z2 = lor.z2[tid];
			weight = lor.weight[tid];
			linevalue = 0.;
	
			ulen2 = powf(x1-x2,2) + powf(y1-y2,2) + powf(z1-z2,2);

			t = ( i * a - x1) / (x2 - x1);
			
			oy = y1 + t * (y2 - y1);
			oz = z1 + t * (z2 - z1);
			
			mlyy = max((int)truncf((oy - (SRTWO * torhw ))/a)+1, 0);
			mhyy = min((int)truncf((oy + (SRTWO * torhw ))/a), ny - 1);
			mlzz = max((int)truncf((oz - (SRTWO * torhw ))/a)+1, 0);
			mhzz = min((int)truncf((oz + (SRTWO * torhw ))/a), nz - 1);

			mlyy = max(mlyy, sharesize * tn1);
			mhyy = min(mhyy, sharesize * (tn1 + 1)-1);
			mlzz = max(mlzz, sharesize * tn2);
			mhzz = min(mhzz, sharesize * (tn2 + 1)-1);
			
			for(int ky = mlyy; ky <= mhyy; ky++)
			{
			    for(int kz = mlzz; kz <= mhzz; kz++)
			    {
					float dy = oy - ky*a, dz = oz - kz*a;
					float inner = dy * (y1-y2) + dz * (z1 - z2);
					float dst2 = dy * dy + dz * dz - inner * inner / ulen2;
					float maxdst2 = torhw * torhw;
			        //dst = sqrtf(pow(oy-ky*a, 2) + powf(oz-kz*a, 2) - powf((oy-ky*a) * (y1-y2) + (oz-kz*a) * (z1-z2), 2) / ulen2);
			        if(dst2 < maxdst2) linevalue += cache[ky -sharesize * tn1 ][ kz - sharesize * tn2] * expf(-dst2/(2.0f * torsgm2)) * weight;
			    }
			}
			lor.linevalue[tid] += linevalue;
			//finish one tile for one lor
			tid += blockDim.x * blockDim.y * gridDim.x;
		}
		__syncthreads();
		}
		}
	}


}

__global__ void xbpro( cudalor lor, float *snmatrix ) 
{
	int nx = d_imageindex[0], ny = d_imageindex[1], nz = d_imageindex[2];
	float a = d_info[0], torhw = d_info[1], torsgm2 = d_info[2];
	int lornum = d_lorindex[0];

	__shared__ float cache[sharesize][sharesize];
	int tid ;
	int cacheIndex1 , cacheIndex2;

	float linevalue, rlinevalue;
	float ulen2, t, oy, oz;
	int mlyy,mhyy,mlzz,mhzz;
	float x1,x2,y1,y2,z1,z2,weight;

	int tilenum1 = (ny + sharesize - 1)/ sharesize, tilenum2 = (nz + sharesize - 1) / sharesize ;

	for(int i=0; i< nx; i++)
	{

        for(int tn1 = 0; tn1 < tilenum1; tn1++)	//for each tile of image. This is due to limited shared memory.
        {
		for(int tn2 = 0; tn2 < tilenum2; tn2++)
		{

        cacheIndex1 = threadIdx.x;

		while(cacheIndex1 < sharesize && ((sharesize * tn1) + cacheIndex1) < ny )	//load a tile of image
        {
		cacheIndex2 = threadIdx.y;
        while(cacheIndex2 < sharesize && ((sharesize * tn2) + cacheIndex2) < nz)
        {
            cache[cacheIndex1][cacheIndex2] = 0.0f;
            cacheIndex2 += blockDim.y;
        }
            cacheIndex1 += blockDim.x;
        }
        __syncthreads();

		tid = threadIdx.x + threadIdx.y * blockDim.x +  blockIdx.x * blockDim.x * blockDim.y;

		while(tid < lornum)
		{
			x1 = lor.x1[tid];
			y1 = lor.y1[tid];
			z1 = lor.z1[tid];
			x2 = lor.x2[tid];
			y2 = lor.y2[tid];
			z2 = lor.z2[tid];
			weight = lor.weight[tid];
	        linevalue = lor.linevalue[tid];
	
	        if(linevalue < ThreshLineValue) rlinevalue = 0.0f;
	        else rlinevalue = 1.0f / linevalue;
	
			ulen2 = powf(x1-x2,2) + powf(y1-y2,2) + powf(z1-z2,2);

			t = ( i * a - x1) / (x2 - x1);
			
			oy = y1 + t * (y2 - y1);
			oz = z1 + t * (z2 - z1);
			
			mlyy = max((int)truncf((oy - (SRTWO * torhw ))/a)+1, 0);
			mhyy = min((int)truncf((oy + (SRTWO * torhw ))/a), ny - 1);
			mlzz = max((int)truncf((oz - (SRTWO * torhw ))/a)+1, 0);
			mhzz = min((int)truncf((oz + (SRTWO * torhw ))/a), nz - 1);

			mlyy = max(mlyy, sharesize * tn1);
			mhyy = min(mhyy, sharesize * (tn1 + 1)-1);
			mlzz = max(mlzz, sharesize * tn2);
			mhzz = min(mhzz, sharesize * (tn2 + 1)-1);
			
			for(int ky = mlyy; ky <= mhyy; ky++)
			{
			    for(int kz = mlzz; kz <= mhzz; kz++)
			    {
					float dy = oy - ky*a, dz = oz - kz*a;
					float inner = dy * (y1-y2) + dz * (z1 - z2);
					float dst2 = dy * dy + dz * dz - inner * inner / ulen2;
					float maxdst2 = torhw * torhw;
			        //dst = sqrtf(pow(oy-ky*a, 2) + powf(oz-kz*a, 2) - powf((oy-ky*a) * (y1-y2) + (oz-kz*a) * (z1-z2), 2) / ulen2);
			        if(dst2 < maxdst2) atomicAdd(&cache[ky -sharesize * tn1 ][ kz - sharesize * tn2], expf(-dst2/(2.0f * torsgm2)) * rlinevalue * weight) ;
			    }
			}

			tid += blockDim.x * blockDim.y * gridDim.x;
		}
		__syncthreads();

		//write the tile of image to global memory
		cacheIndex1 = threadIdx.x;

		while(cacheIndex1 < sharesize && ((sharesize * tn1) + cacheIndex1) < ny )	//load a tile of image
        {
        cacheIndex2 = threadIdx.y;
        while(cacheIndex2 < sharesize && ((sharesize * tn2) + cacheIndex2) < nz)
        {
            atomicAdd(&snmatrix[i  + ((sharesize * tn1) + cacheIndex1) * nx + ((sharesize * tn2) + cacheIndex2) * nx * ny], cache[cacheIndex1][cacheIndex2]);
            cacheIndex2 += blockDim.y;
        }
            cacheIndex1 += blockDim.x;
        }
		__syncthreads();

		}
		}
	}


}


//add f to functions such as expf(), powf(), which stands for single precision. reduce some redundant variables. the trouble comes from cache[][], need to move cacheIndex2 increment out of while.some cores not launched due to tid limit.
__global__ void yfpro( cudalor lor, float *smatrix ) 
{

    int nx = d_imageindex[0], ny =d_imageindex[1], nz = d_imageindex[2];
    float a = d_info[0], torhw = d_info[1], torsgm2 = d_info[2];
	int lornum = d_lorindex[1];

	__shared__ float cache[sharesize][sharesize];
	int tid;
	int cacheIndex1, cacheIndex2;

    float linevalue = 0.;
    float ulen2, t, ox,oz;
    int mlxx,mhxx,mlzz,mhzz;
    float x1,x2,y1,y2,z1,z2,weight;

	int tilenum1 = (nx + sharesize - 1)/ sharesize, tilenum2 = (nz + sharesize - 1) / sharesize ;

//	testmatrix[0] = 20.0f;

    for(int i=0; i< ny; i++)
    {

        for(int tn1 = 0; tn1 < tilenum1; tn1++)	//for each tile of image. This is due to limited shared memory.
        {
		for(int tn2 = 0; tn2 < tilenum2; tn2++)
		{

        cacheIndex1 = threadIdx.x;

		while(cacheIndex1 < sharesize && ((sharesize * tn1) + cacheIndex1) < nx )	//load a tile of image
        {
		cacheIndex2 = threadIdx.y;
        while(cacheIndex2 < sharesize && ((sharesize * tn2) + cacheIndex2) < nz)
        {

            cache[cacheIndex1][cacheIndex2] = smatrix[((sharesize * tn1) + cacheIndex1) + i * nx + ((sharesize * tn2) + cacheIndex2)* nx * ny];
//			if(tn1 == 0 && tn2 == 0 && i == 0) testmatrix[cacheIndex1*sharesize + cacheIndex2] = threadIdx.y;
			cacheIndex2 += blockDim.y;

        }
            cacheIndex1 += blockDim.x;
        }
        __syncthreads();

		tid = threadIdx.x + threadIdx.y * blockDim.x +  blockIdx.x * blockDim.x * blockDim.y;

	    while(tid < lornum)
	    {
	        x1 = lor.x1[tid];
	        y1 = lor.y1[tid];
	        z1 = lor.z1[tid];
	        x2 = lor.x2[tid];
	        y2 = lor.y2[tid];
	        z2 = lor.z2[tid];
			weight = lor.weight[tid];
			linevalue = 0.;
	
	        ulen2 = powf(x1-x2,2) + powf(y1-y2,2) + powf(z1-z2,2);


            t = ( i * a - y1) / (y2 - y1);

            ox = x1 + t * (x2 - x1);
            oz = z1 + t * (z2 - z1);

			mlxx = max((int)truncf((ox - (SRTWO * torhw ))/a)+1, 0);
			mhxx = min((int)truncf((ox + (SRTWO * torhw ))/a), nx - 1);
			mlzz = max((int)truncf((oz - (SRTWO * torhw ))/a)+1, 0);
			mhzz = min((int)truncf((oz + (SRTWO * torhw ))/a), nz - 1);

			mlxx = max(mlxx, sharesize * tn1);
			mhxx = min(mhxx, sharesize * (tn1 + 1)-1);
			mlzz = max(mlzz, sharesize * tn2);
			mhzz = min(mhzz, sharesize * (tn2 + 1)-1);
			                                                                                                                                                                          
            for(int kx = mlxx; kx <= mhxx; kx++)
            {
                for(int kz = mlzz; kz <= mhzz; kz++)
                {
					float dx = ox - kx*a, dz = oz - kz*a;
					float inner = dx * (x1-x2) + dz * (z1 - z2);
					float dst2 = dx * dx + dz * dz - inner * inner / ulen2;
					float maxdst2 = torhw * torhw;
					//dst = sqrtf(powf(ox-kx*a, 2) + powf(oz-kz*a, 2) - powf((ox-kx*a) * (x1-x2) + (oz-kz*a) * (z1-z2), 2) / ulen2);
					if(dst2 < maxdst2) linevalue += cache[kx -sharesize * tn1 ][ kz - sharesize * tn2] * expf(-dst2/(2.0f * torsgm2)) * weight;
				}
            }

    		lor.linevalue[tid] += linevalue;
    		//finish one tile for one lor
	        tid += blockDim.x * blockDim.y * gridDim.x;
	    }
		//finish one tile for all lors
		__syncthreads();

        }
        }
		//finish all tiles in a slice
    }
	//finish all slices


}

__global__ void ybpro( cudalor lor, float *snmatrix )
{
    int nx = d_imageindex[0], ny =d_imageindex[1], nz = d_imageindex[2];
    float a = d_info[0], torhw = d_info[1], torsgm2 = d_info[2];
	int lornum = d_lorindex[1];

	__shared__ float cache[sharesize][sharesize];
	int tid ;
	int cacheIndex1 , cacheIndex2 ;

    float linevalue , rlinevalue;
    float ulen2, t, ox,oz;
    int mlxx,mhxx,mlzz,mhzz;
    float x1,x2,y1,y2,z1,z2,weight;

	int tilenum1 = (nx + sharesize - 1)/ sharesize, tilenum2 = (nz + sharesize - 1) / sharesize ;


    for(int i=0; i< ny; i++)
    {
        for(int tn1 = 0; tn1 < tilenum1; tn1++)	//for each tile of image. This is due to limited shared memory.
        {
		for(int tn2 = 0; tn2 < tilenum2; tn2++)
		{

		//initialize a tile of image in shared memory to zero.
        cacheIndex1 = threadIdx.x;
		
		while(cacheIndex1 < sharesize && ((sharesize * tn1) + cacheIndex1) < nx )	
        {
		cacheIndex2 = threadIdx.y;
        while(cacheIndex2 < sharesize && ((sharesize * tn2) + cacheIndex2) < nz)
        {
            cache[cacheIndex1][cacheIndex2] = 0.0f ;
            cacheIndex2 += blockDim.y;
        }
            cacheIndex1 += blockDim.x;
        }
        __syncthreads();

		tid = threadIdx.x + threadIdx.y * blockDim.x +  blockIdx.x * blockDim.x * blockDim.y;

		//calculate contribution of all lors to the tile of image
	    while(tid < lornum)
	    {
	        linevalue = lor.linevalue[tid];
	
	        if(linevalue < ThreshLineValue) rlinevalue = 0.0f;
	        else rlinevalue = 1.0f / linevalue;
	
	        x1 = lor.x1[tid];
	        y1 = lor.y1[tid];
	        z1 = lor.z1[tid];
	        x2 = lor.x2[tid];
	        y2 = lor.y2[tid];
	        z2 = lor.z2[tid];
			weight = lor.weight[tid];
	
	        ulen2 = powf(x1-x2,2) + powf(y1-y2,2) + powf(z1-z2,2);
	
            t = ( i * a - y1) / (y2 - y1);

            //if(t<0. || t> 1.) continue;

            ox = x1 + t * (x2 - x1);
            oz = z1 + t * (z2 - z1);

			mlxx = max((int)truncf((ox - (SRTWO * torhw ))/a)+1, 0);
			mhxx = min((int)truncf((ox + (SRTWO * torhw ))/a), nx - 1);
			mlzz = max((int)truncf((oz - (SRTWO * torhw ))/a)+1, 0);
			mhzz = min((int)truncf((oz + (SRTWO * torhw ))/a), nz - 1);

			mlxx = max(mlxx, sharesize * tn1);
			mhxx = min(mhxx, sharesize * (tn1 + 1)-1);
			mlzz = max(mlzz, sharesize * tn2);
			mhzz = min(mhzz, sharesize * (tn2 + 1)-1);

            for(int kx = mlxx; kx <= mhxx; kx++)
            {
                for(int kz = mlzz; kz <= mhzz; kz++)
                {
					float dx = ox - kx*a, dz = oz - kz*a;
					float inner = dx * (x1-x2) + dz * (z1 - z2);
					float dst2 = dx * dx + dz * dz - inner * inner / ulen2;
					float maxdst2 = torhw * torhw;
                    //dst = sqrtf(powf(ox-kx*a, 2) + powf(oz-kz*a, 2) - powf((ox-kx*a) * (x1-x2) + (oz-kz*a) * (z1-z2), 2) / ulen2);
                    if(dst2 < maxdst2) atomicAdd(&cache[kx -sharesize * tn1 ][ kz - sharesize * tn2], expf(-dst2/(2.0f * torsgm2)) * rlinevalue * weight) ;

                }
            }
	    
			tid += blockDim.x * blockDim.y * gridDim.x;
		}

        __syncthreads();

		//add the tile of image to global memory
		cacheIndex1 = threadIdx.x;
		
		while(cacheIndex1 < sharesize && ((sharesize * tn1) + cacheIndex1) < nx )	//load a tile of image
		{
		cacheIndex2 = threadIdx.y;
		while(cacheIndex2 < sharesize && ((sharesize * tn2) + cacheIndex2) < nz)
		{
		    atomicAdd(&snmatrix[((sharesize * tn1)  + cacheIndex1)  + i * nx + ((sharesize * tn2) + cacheIndex2) * nx * ny ],cache[cacheIndex1][cacheIndex2]);
		    cacheIndex2 += blockDim.y;
		}
		    cacheIndex1 += blockDim.x;
		}
		__syncthreads();

        }
        }
    }


}


__global__ void zfpro( cudalor lor, float *smatrix)
{
    int nx = d_imageindex[0], ny = d_imageindex[1], nz = d_imageindex[2];
    float a = d_info[0], torhw = d_info[1], torsgm2 = d_info[2];
	int lornum = d_lorindex[2];

	__shared__ float cache[sharesize][sharesize];
	int tid ;
	int cacheIndex1 , cacheIndex2 ;

    float linevalue = 0.;
    float ulen2, t, ox, oy;
    int mlxx,mhxx,mlyy,mhyy;
    float x1,x2,y1,y2,z1,z2,weight;

	int tilenum1 = (nx + sharesize - 1)/ sharesize, tilenum2 = (ny + sharesize - 1) / sharesize ;

    for(int i=0; i< nz; i++)
    {

        for(int tn1 = 0; tn1 < tilenum1; tn1++)	//for each tile of image. This is due to limited shared memory.
        {
		for(int tn2 = 0; tn2 < tilenum2; tn2++)
		{

        cacheIndex1 = threadIdx.x;

		while(cacheIndex1 < sharesize && ((sharesize * tn1) + cacheIndex1) < nx )	//load a tile of image
        {
		cacheIndex2 = threadIdx.y;
        while(cacheIndex2 < sharesize && ((sharesize * tn2) + cacheIndex2) < ny)
        {
            cache[cacheIndex1][cacheIndex2] = smatrix[((sharesize * tn1) + cacheIndex1) + ((sharesize * tn2) + cacheIndex2) * nx + i * nx * ny];
            cacheIndex2 += blockDim.y;
        }
            cacheIndex1 += blockDim.x;
        }
      	__syncthreads();

		tid = threadIdx.x + threadIdx.y * blockDim.x +  blockIdx.x * blockDim.x * blockDim.y;

	    while(tid < lornum)
	    {
	        x1 = lor.x1[tid];
	        y1 = lor.y1[tid];
	        z1 = lor.z1[tid];
	        x2 = lor.x2[tid];
	        y2 = lor.y2[tid];
	        z2 = lor.z2[tid];
			weight = lor.weight[tid];
			linevalue = 0.;
	
	        ulen2 = powf(x1-x2,2) + powf(y1-y2,2) + powf(z1-z2,2);

            t = ( i * a - z1) / (z2 - z1);

            oy = y1 + t * (y2 - y1);
            ox = x1 + t * (x2 - x1);

            mlyy = max((int)truncf((oy - (SRTWO * torhw ))/a)+1, 0);
            mhyy = min((int)truncf((oy + (SRTWO * torhw ))/a), ny - 1);
            mlxx = max((int)truncf((ox - (SRTWO * torhw ))/a)+1, 0);
            mhxx = min((int)truncf((ox + (SRTWO * torhw ))/a), nx - 1);

			mlxx = max(mlxx, sharesize * tn1);
			mhxx = min(mhxx, sharesize * (tn1 + 1)-1);
			mlyy = max(mlyy, sharesize * tn2);
			mhyy = min(mhyy, sharesize * (tn2 + 1)-1);

            for(int kx = mlxx; kx <= mhxx; kx++)
            {
                for(int ky = mlyy; ky <= mhyy; ky++)
                {
					float dy = oy - ky*a, dx = ox - kx*a;
					float inner = dy * (y1-y2) + dx * (x1 - x2);
					float dst2 = dy * dy + dx * dx - inner * inner / ulen2;
					float maxdst2 = torhw * torhw;
                    //dst = sqrtf(powf(oy-ky*a, 2) + powf(ox-kx*a, 2) - powf((oy-ky*a) * (y1-y2) + (ox-kx*a) * (x1-x2), 2) / ulen2);
                    if(dst2 < maxdst2) linevalue += cache[kx -sharesize * tn1 ][ ky - sharesize * tn2] * expf(-dst2/(2.0f * torsgm2)) * weight;
                }
            }
	        lor.linevalue[tid] += linevalue;
    	    //finish one tile for one lor
	       	tid += blockDim.x * blockDim.y * gridDim.x;
		}

        __syncthreads();
        }
        }
    }

  
}


__global__ void zbpro( cudalor lor, float *snmatrix )
{
    int nx = d_imageindex[0], ny = d_imageindex[1], nz = d_imageindex[2];
    float a = d_info[0], torhw = d_info[1], torsgm2 = d_info[2];
	int lornum = d_lorindex[2];

	__shared__ float cache[sharesize][sharesize];
	int tid ;
	int cacheIndex1 , cacheIndex2 ;

    float linevalue, rlinevalue;
    float ulen2, t, ox, oy;
    int mlxx,mhxx,mlyy,mhyy;
    float x1,x2,y1,y2,z1,z2,weight;

	int tilenum1 = (nx + sharesize - 1)/ sharesize, tilenum2 = (ny + sharesize - 1) / sharesize ;

    for(int i=0; i< nz; i++)
    {

        for(int tn1 = 0; tn1 < tilenum1; tn1++)	//for each tile of image. This is due to limited shared memory.
        {
		for(int tn2 = 0; tn2 < tilenum2; tn2++)
		{

        cacheIndex1 = threadIdx.x;

		while(cacheIndex1 < sharesize && ((sharesize * tn1) + cacheIndex1) < nx )	//load a tile of image
        {
		cacheIndex2 = threadIdx.y;
        while(cacheIndex2 < sharesize && ((sharesize * tn2) + cacheIndex2) < ny)
        {
            cache[cacheIndex1][cacheIndex2] = 0.0f;
            cacheIndex2 += blockDim.y;
        }
            cacheIndex1 += blockDim.x;
        }
      	__syncthreads();

		tid = threadIdx.x + threadIdx.y * blockDim.x +  blockIdx.x * blockDim.x * blockDim.y;

	    while(tid < lornum)
	    {
	        x1 = lor.x1[tid];
	        y1 = lor.y1[tid];
	        z1 = lor.z1[tid];
	        x2 = lor.x2[tid];
	        y2 = lor.y2[tid];
	        z2 = lor.z2[tid];
			weight = lor.weight[tid];
	        linevalue = lor.linevalue[tid];
	
	        if(linevalue < ThreshLineValue) rlinevalue = 0.0f;
	        else rlinevalue = 1.0f / linevalue;
	
	        ulen2 = powf(x1-x2,2) + powf(y1-y2,2) + powf(z1-z2,2);

            t = ( i * a - z1) / (z2 - z1);

            oy = y1 + t * (y2 - y1);
            ox = x1 + t * (x2 - x1);

            mlyy = max((int)truncf((oy - (SRTWO * torhw ))/a)+1, 0);
            mhyy = min((int)truncf((oy + (SRTWO * torhw ))/a), ny - 1);
            mlxx = max((int)truncf((ox - (SRTWO * torhw ))/a)+1, 0);
            mhxx = min((int)truncf((ox + (SRTWO * torhw ))/a), nx - 1);

			mlxx = max(mlxx, sharesize * tn1);
			mhxx = min(mhxx, sharesize * (tn1 + 1)-1);
			mlyy = max(mlyy, sharesize * tn2);
			mhyy = min(mhyy, sharesize * (tn2 + 1)-1);

            for(int kx = mlxx; kx <= mhxx; kx++)
            {
                for(int ky = mlyy; ky <= mhyy; ky++)
                {
					float dy = oy - ky*a, dx = ox - kx*a;
					float inner = dy * (y1-y2) + dx * (x1 - x2);
					float dst2 = dy * dy + dx * dx - inner * inner / ulen2;
					float maxdst2 = torhw * torhw;
                    //dst = sqrtf(powf(oy-ky*a, 2) + powf(ox-kx*a, 2) - powf((oy-ky*a) * (y1-y2) + (ox-kx*a) * (x1-x2), 2) / ulen2);
                    if(dst2 < maxdst2) atomicAdd(&cache[kx -sharesize * tn1 ][ ky - sharesize * tn2], expf(-dst2/(2.0f * torsgm2)) * rlinevalue * weight) ;
                }
            }

	       	tid += blockDim.x * blockDim.y * gridDim.x;
		}

        __syncthreads();

		//write the tile of image to global memory
		cacheIndex1 = threadIdx.x;
		
		while(cacheIndex1 < sharesize && ((sharesize * tn1) + cacheIndex1) < nx )	//load a tile of image
		{
		cacheIndex2 = threadIdx.y;
		while(cacheIndex2 < sharesize && ((sharesize * tn2) + cacheIndex2) < ny)
		{
		    atomicAdd(&snmatrix[((sharesize * tn1) + cacheIndex1) + ((sharesize * tn2) + cacheIndex2) * nx + i * nx * ny], cache[cacheIndex1][cacheIndex2]);
		    cacheIndex2 += blockDim.y;
		}
		    cacheIndex1 += blockDim.x;
		}
		
		__syncthreads();

        }
        }
    }

  
}


int main(int argc, char* argv[])
{
	vector<string> vpara;
	string line;
	stringstream ss;
	ifstream config ("configRecon.txt");
    int itenum; //number of iterations.
	int startDiff = 1;

	if (config.is_open())
	{
		while ( getline (config,line) )
		{
			vpara=explode(line,' ');
			if(vpara[0]=="FOV") {ss<<vpara[2];ss>>bndry[0];ss.clear();ss<<vpara[3];ss>>bndry[1];ss.clear();ss<<vpara[4];ss>>bndry[2];ss.clear();} else
			if(vpara[0]=="GridSize") {ss<<vpara[2];ss>>a;ss.clear();} else
			if(vpara[0]=="TorHalfWidth") {ss<<vpara[2];ss>>torhw;ss.clear();} else
			if(vpara[0]=="TorSigma") {ss<<vpara[2];ss>>torsgm;ss.clear();} else
			if(vpara[0]=="NumberOfIterations") {ss<<vpara[2];ss>>itenum;ss.clear();} else
			if(vpara[0]=="Regularization") {ss<<vpara[2];ss>>rgl;ss.clear();} else
			if(vpara[0]=="Normalization") {ss<<vpara[2];ss>>norma;ss.clear();} else
			if(vpara[0]=="ThreshNorm") {ss<<vpara[2];ss>>ThreshNorm;ss.clear();} else
			if(vpara[0]=="BetaR") {ss<<vpara[2];ss>>beta;ss.clear();} else
			if(vpara[0]=="BlurR") {ss<<vpara[2];ss>>blur;ss.clear();} else
			if(vpara[0]=="XsigmaRB") {ss<<vpara[2];ss>>bsgm[0];ss.clear();} else
			if(vpara[0]=="YsigmaRB") {ss<<vpara[2];ss>>bsgm[1];ss.clear();} else
			if(vpara[0]=="ZsigmaRB") {ss<<vpara[2];ss>>bsgm[2];ss.clear();} else
			if(vpara[0]=="Weight") {ss<<vpara[2];ss>>wgt;ss.clear();}

		}
		config.close();
	}
	else cout << "Unable to open config file"<<endl;

	cout<<"-------------------------------------------"<<endl;
	cout<<"Input parameters:"<<endl;
	cout<<"FOV: "<<bndry[0]<<" mm x "<<bndry[1]<<" mm x "<<bndry[2]<<" mm"<<endl;
	cout<<"Grid size: "<<a<<" mm"<<endl;
	cout<<"TOR half width: "<<torhw<<" mm"<<endl;
	cout<<"TOR sigma: "<<torsgm<<" mm"<<endl;
	cout<<"Number of iterations: "<<itenum<<endl;
	cout<<"Normalization?: "<<norma<<endl;
	if(norma != 0) cout<<"ThreshNorm: "<<ThreshNorm<<endl;
	cout<<"Regularization?: "<<rgl<<endl;
	cout<<"Include weight?: "<<wgt<<endl;
	if(rgl==1)
	{
		cout<<"Beta for regularization: "<<beta<<endl;
		cout<<"Blur?: "<<blur<<endl;
        if(blur==1)
        {
            cout<<"Xsigma for blur: "<<bsgm[0]<<" mm"<<endl;
            cout<<"Ysigma for blur: "<<bsgm[1]<<" mm"<<endl;
            cout<<"Zsigma for blur: "<<bsgm[2]<<" mm"<<endl;
			for(int i=0; i<3; i++) {rads[i] = bsgm[i] * sqrt(-2. * log (bthresh)); indr[i] = trunc(rads[i]/a);}
			hipMemcpyToSymbol(HIP_SYMBOL(d_bsgm), bsgm, 3 * sizeof(float), 0, hipMemcpyHostToDevice);
			hipMemcpyToSymbol(HIP_SYMBOL(d_rads), rads, 3 * sizeof(float), 0, hipMemcpyHostToDevice); 
			hipMemcpyToSymbol(HIP_SYMBOL(d_indr), indr, 3 * sizeof(int), 0, hipMemcpyHostToDevice); 
        }
		
	}
	else beta = 0.;
	cout<<"-------------------------------------------"<<endl;




	//float stp;
	//vector<float> poimage;  //prior image
	float buff2;
	//float sumpop=0., sumimp=0.;
	ifstream finrgl;
	ifstream finlast;
    string rglname;
    string lastname;

	dim3 threads(threadsperBlock, threadsperBlock);
    nummainaxis = (int*) malloc(3 * sizeof(int));
	//cudalor dev_xlor, dev_ylor, dev_zlor;   //define variables for cudalor.

	msize = ceil(bndry[0] / a) * ceil(bndry[1] / a) *ceil( bndry[2] / a);
	int nx = ceil(bndry[0] / a);
	int ny = ceil(bndry[1] / a);
	int nz = ceil( bndry[2] / a);

//copy fundamental variables to cuda.
	int *temp_imageindex;
	temp_imageindex = (int*) malloc(4 * sizeof(int));
	temp_imageindex[0] = nx;
	temp_imageindex[1] = ny;
	temp_imageindex[2] = nz;
	temp_imageindex[3] = msize;

	float *temp_info;
	temp_info = (float*) malloc(4 * sizeof(float));
	temp_info[0] = a;
	temp_info[1] = torhw;
	temp_info[2] = pow(torsgm,2);	//for higher efficiency in gpu
	temp_info[3] = beta;
	
	//int *dev_imageindex, *dev_lorindex;
	//float *dev_info;
	//hipMalloc((void**) &dev_imageindex, 4*sizeof(int) );
	//hipMalloc((void**) &dev_lorindex, 3*sizeof(int) );
	//hipMalloc((void**) &dev_info, 5*sizeof(float) );

//	cout<<"before copying to constant memory"<<endl;
	hipMemcpyToSymbol(HIP_SYMBOL(d_imageindex), temp_imageindex, 4 * sizeof(int), 0, hipMemcpyHostToDevice);
	//hipMemcpy(dev_imageindex, temp_imageindex, 4 * sizeof(int), hipMemcpyHostToDevice);
//	hipMemcpy(dev_lorindex, nummainaxis, 3 * sizeof(int), hipMemcpyHostToDevice); //should be after taking the input data
	hipMemcpyToSymbol(HIP_SYMBOL(d_info), temp_info, 4 * sizeof(float), 0, hipMemcpyHostToDevice);
	//hipMemcpy(dev_info, temp_info,  5* sizeof(float), hipMemcpyHostToDevice);	//should be after getting maxnorm and beta.

	//delete [] temp_imageindex;
	//delete [] temp_info;



	ifstream fnorm;	//use this if input is normalization image
	string filenorm;
	ofstream fnormout;
	float *lineval;	//used for initializing linevalue of cudalor. As hipMemset deals with each byte, not each floating point number.
	float *normpara;
	normpara = (float*) malloc(2 * sizeof(float));
	float maxnorm;
	normpara[1] = ThreshNorm;

	if(norma == 1)	//generate sensitivity image from backprojecting all possible LORs or simulation data with weight being 1
	{
		filenorm = argv[4];
		cout<<"Sorting LORs for normalization and copying to device memory......"<<endl;
		preplor(filenorm, 1); //  read lors in the file, sort lors, copy to cud
        cout<<"Normalization: Number of LORs in each main axis (x,y,x): "<<nummainaxis[0]<<" "<<nummainaxis[1]<<" "<<nummainaxis[2]<<endl;
        //hipMemcpy(dev_lorindex, nummainaxis, 3 * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(d_lorindex), nummainaxis, 3 * sizeof(int), 0, hipMemcpyHostToDevice);
		
		normimage = (float*) malloc(msize * sizeof(float));
		hipMalloc((void**) &dev_normimage, msize*sizeof(float) );
		hipMemset( dev_normimage, 0.0f, msize*sizeof(float));
	
		timinggpu.StartCounter();
		lineval = (float*) malloc(nummainaxis[0]*sizeof(float));
		for(int ii=0; ii<nummainaxis[0]; ii++) lineval[ii] = 1.0f;
		hipMemcpy(dev_xlor.linevalue, lineval, nummainaxis[0]*sizeof(float), hipMemcpyHostToDevice);
		lineval = (float*) malloc(nummainaxis[1]*sizeof(float));
		for(int ii=0; ii<nummainaxis[1]; ii++) lineval[ii] = 1.0f;
		hipMemcpy(dev_ylor.linevalue, lineval, nummainaxis[1]*sizeof(float), hipMemcpyHostToDevice);  
		lineval = (float*) malloc(nummainaxis[2]*sizeof(float));
		for(int ii=0; ii<nummainaxis[2]; ii++) lineval[ii] = 1.0f;
		hipMemcpy(dev_zlor.linevalue, lineval, nummainaxis[2]*sizeof(float), hipMemcpyHostToDevice);  

		free(lineval);
		timeall.memoryIO += timinggpu.GetCounter();

		xbpro<<<blocksPerGrid, threads>>>(dev_xlor, dev_normimage);
		ybpro<<<blocksPerGrid, threads>>>(dev_ylor, dev_normimage);
		zbpro<<<blocksPerGrid, threads>>>(dev_zlor, dev_normimage);

		timinggpu.StartCounter();
		hipMemcpy(normimage, dev_normimage, msize*sizeof(float), hipMemcpyDeviceToHost);
		timeall.memoryIO += timinggpu.GetCounter();
		//hipMemcpy(ylor.linevalue, dev_ylor.linevalue, nummainaxis[1]*sizeof(float), hipMemcpyDeviceToHost);
		//cout<<ylor.linevalue[0]<<endl;
		
		fnormout.open("normImage", ios::out | ios::binary);
		maxnorm = 0.0;
		for(int iii=0; iii< msize; iii++)
		{
			fnormout.write( (char*)&normimage[iii], sizeof(float));
			if(maxnorm < normimage[iii]) maxnorm = normimage[iii];
		}
		fnormout.close();
		normpara[0] = maxnorm;
		hipMemcpyToSymbol(HIP_SYMBOL(d_norm), normpara,  2 * sizeof(float), 0, hipMemcpyHostToDevice);
		cout<<"Finish creating normalization image."<<endl;

	}

	else if(norma == 2)	//read sensitivity image from input file.
	{
		filenorm = argv[4];
		cout<<"Reading normalization image......"<<endl;
		fnorm.open(filenorm.c_str(), ios::in | ios::binary);
		if (fnorm.is_open()){
			normimage = (float*) malloc(msize * sizeof(float));
			maxnorm = 0.0;
			for(int iii=0; iii< msize; iii++)
			{
				fnorm.read( (char*)&normimage[iii], sizeof(float));
				if(maxnorm < normimage[iii]) maxnorm = normimage[iii];
			}
		}
		else cout<<"Unable to open normImage file!!"<<endl;
		fnorm.close();
		normpara[0] = maxnorm;

		timinggpu.StartCounter();
		hipMalloc((void**) &dev_normimage, msize*sizeof(float) );
		hipMemcpy(dev_normimage, normimage, msize*sizeof(float), hipMemcpyHostToDevice);
		timeall.memoryIO += timinggpu.GetCounter();

		hipMemcpyToSymbol(HIP_SYMBOL(d_norm), normpara, 2 *  sizeof(float), 0, hipMemcpyHostToDevice);
		cout<<"Finish reading normalization image."<<endl;
	}


	else if(norma == 3)	//generate sensitivity image from backprojecting simulation data with different weight
	{
		filenorm = argv[4];
		cout<<"Sorting LORs for normalization and copying to device memory......"<<endl;
		preplor(filenorm, 1); //  read lors in the file, sort lors, copy to cud
        cout<<"Normalization: Number of LORs in each main axis (x,y,x): "<<nummainaxis[0]<<" "<<nummainaxis[1]<<" "<<nummainaxis[2]<<endl;
        //hipMemcpy(dev_lorindex, nummainaxis, 3 * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(d_lorindex), nummainaxis, 3 * sizeof(int), 0, hipMemcpyHostToDevice);
		
		normimage = (float*) malloc(msize * sizeof(float));
		hipMalloc((void**) &dev_normimage, msize*sizeof(float) );
		hipMemset( dev_normimage, 0, msize*sizeof(float)); 

		hipMemset( dev_xlor.linevalue, 0, nummainaxis[0]*sizeof(float));	// improved normalization method
		hipMemset( dev_ylor.linevalue, 0, nummainaxis[1]*sizeof(float));
		hipMemset( dev_zlor.linevalue, 0, nummainaxis[2]*sizeof(float));
		float *normphantom = (float*) malloc(msize * sizeof(float)), *dev_normphantom;
		hipMalloc((void**) &dev_normphantom, msize * sizeof(float));
		for(int k=0; k< nz; k++){
			for(int j=0; j< ny; j++){
				for(int i=0; i< nx; i++){
					float cx = i * a - (bndry[0]/2 - 0.5 * a);
					float cy = j * a - (bndry[1]/2 - 0.5 * a);
					float cz = k * a - (bndry[2]/2 - 0.5 * a);
					float temp = 0.0f;
					if(cx * cx + cy * cy < 3025.0 && cz >= -63 && cz <= 63) temp = 0.01f;
					//if(cz >= -63 && cz <= 63) temp = 0.001f;
					normphantom[i + j * nx + k * nx * ny] = temp;

				}
			}
		}
		hipMemcpy(dev_normphantom, normphantom, msize*sizeof(float), hipMemcpyHostToDevice);
		
		xfpro<<<blocksPerGrid, threads>>>(dev_xlor, dev_normphantom);
		yfpro<<<blocksPerGrid, threads>>>(dev_ylor, dev_normphantom);
		zfpro<<<blocksPerGrid, threads>>>(dev_zlor, dev_normphantom);
		xbpro<<<blocksPerGrid, threads>>>(dev_xlor, dev_normimage);
		ybpro<<<blocksPerGrid, threads>>>(dev_ylor, dev_normimage);
		zbpro<<<blocksPerGrid, threads>>>(dev_zlor, dev_normimage);

		timinggpu.StartCounter();
		hipMemcpy(normimage, dev_normimage, msize*sizeof(float), hipMemcpyDeviceToHost);
		timeall.memoryIO += timinggpu.GetCounter();
		//hipMemcpy(ylor.linevalue, dev_ylor.linevalue, nummainaxis[1]*sizeof(float), hipMemcpyDeviceToHost);
		//cout<<ylor.linevalue[0]<<endl;
		
		fnormout.open("normImage", ios::out | ios::binary);
		maxnorm = 0.0;
		for(int iii=0; iii< msize; iii++)
		{
			fnormout.write( (char*)&normimage[iii], sizeof(float));
			if(maxnorm < normimage[iii]) maxnorm = normimage[iii];
		}
		fnormout.close();
		normpara[0] = maxnorm;
		hipMemcpyToSymbol(HIP_SYMBOL(d_norm), normpara,  2 * sizeof(float), 0, hipMemcpyHostToDevice);
		cout<<"Finish creating normalization image."<<endl;

	}

	else if(norma == 0) {}

	else cout<<"Unkown indicator for normalization option!!"<<endl;




// open file that contains input lors for image reconstruction. This should be after normalization file read.
	//ifstream fin;
	string filein=argv[1];
	//fin.open(filein.c_str());
	cout<<"Sorting LORs and copying to device memory......"<<endl;
    preplor(filein, 0); //  read lors in the file, sort lors, copy to cuda
	cout<<"Finish sorting and copying."<<endl;
	//hipMemcpy(dev_lorindex, nummainaxis, 3 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_lorindex), nummainaxis, 3 * sizeof(int), 0, hipMemcpyHostToDevice);
	//fin.close();


	timing.StartCounter();
	smatrix = (float*) malloc(msize * sizeof(float));
	snmatrix = (float*) malloc(msize * sizeof(float));
	poimage = (float*) malloc(msize * sizeof(float));
	lastimage = (float*) malloc(msize * sizeof(float));
	bmatrix = (float*) malloc(msize * sizeof(float));
    //deri = (float*) malloc(msize * sizeof(float));
	allweight = (float*) malloc(msize * sizeof(float));
	timeall.memoryIO += timing.GetCounter();

//	cout<<"after feeding data to cudalor"<<endl;
	if(rgl == 1){
		rglname=argv[3];
		finrgl.open(rglname.c_str(), ios::in | ios::binary);
	
		//read prior image into ram
        //sumpop = 0.0;
		if (finrgl.is_open()){
			for(int i=0; i< msize; i++)
			{
				finrgl.read((char *)&poimage[i],sizeof(buff2));
				//poimage[i] = buff2;
				//sumpop += buff2;
			}
		}
		else cout<<"Unable to open rgl image!!"<<endl;
		finrgl.close();

		lastname=argv[5];
		finlast.open(lastname.c_str(), ios::in | ios::binary);
		if (finlast.is_open()){
			for(int i=0; i< msize; i++)
			{
				finlast.read((char *)&lastimage[i],sizeof(buff2));
			}
		}
		else cout<<"Unable to open rgl image!!"<<endl;
		finlast.close();
	}


//re-define beta, beta_new = beta * A / 2. Also initialize smatrix.
	if(norma == 0) {
		beta = beta * float(numline)/msize / 2.0f;
		for(int i=0; i<msize; i++) smatrix[i] = float(numline)/msize;
	}
	else {
		float sumNormimage = 0.0f;
		for(int i=0; i< msize; i++) sumNormimage += normimage[i];
		sumNormimage = sumNormimage / maxnorm;
		beta = beta * float(numline)/sumNormimage / 2.0f;
		if(startDiff == true){	
			for(int i=0; i<msize; i++) {
				smatrix[i] = lastimage[i];
			}
		}
		else {
			for(int i=0; i<msize; i++) 
				smatrix[i] = float(numline)/sumNormimage;
		}
	}

	temp_info[3] = beta;
	hipMemcpyToSymbol(HIP_SYMBOL(d_info), temp_info, 4 * sizeof(float), 0, hipMemcpyHostToDevice);
	
	//set aves for scaling. In this case, it remains unchanged across iterations.
	float allave = smatrix[0];
	hipMemcpyToSymbol(HIP_SYMBOL(aves), &allave, sizeof(float), 0, hipMemcpyHostToDevice);
	float hostAve = allave; //for storing the value of A.


//	cout<<"after cuda memory copy"<<endl;

	float *dev_smatrix, *dev_snmatrix, *dev_poimage, *dev_deri, *dev_bmatrix, *dev_allweight;

	timinggpu.StartCounter();
	hipMalloc((void**) &dev_smatrix, msize*sizeof(float) );
	hipMalloc((void**) &dev_snmatrix, msize*sizeof(float) );
	if(rgl == 1)
	{
		hipMalloc((void**) &dev_poimage, msize*sizeof(float) );
		//hipMalloc((void**) &dev_deri, msize*sizeof(float) );
		hipMalloc((void**) &dev_bmatrix, msize*sizeof(float) );
		hipMemcpy( dev_poimage, poimage, msize*sizeof(float), hipMemcpyHostToDevice );
		if(blur == 1)
		{
			hipMalloc((void**) &dev_bmatrix, msize*sizeof(float) );
			hipMalloc((void**) &dev_allweight, msize*sizeof(float) );
		}
	}
	timeall.memoryIO += timinggpu.GetCounter();


	float rgimg, sumi, sumde2, sumde;
	//int idxy;
	//int ci,cj,ck,li,hi,lj,hj,lk,hk;

	//float senratio = 0.0;

//	cout<<"before iteration"<<endl;

//	float *test, *dev_test;
//	test = (float*) malloc( sizeof(float));
//	
//	hipMalloc( (void**)&dev_test, sizeof(float) );

//	float *testmatrix, *dev_testmatrix;
//	hipMalloc((void**) &dev_testmatrix, sharesize * sharesize *sizeof(float) );
//	hipMemset( dev_testmatrix, 0.0f, sharesize * sharesize * sizeof(float));
//	testmatrix = (float *) malloc(sharesize * sharesize * sizeof(float));


	cout<<"Number of LORs in each main axis (x,y,x): "<<nummainaxis[0]<<" "<<nummainaxis[1]<<" "<<nummainaxis[2]<<endl;

	float *gave, *dev_gave;
    int *gnumave, *dev_gnumave;
	gave = (float*) malloc(blocksPerGrid * sizeof(float));
	hipMalloc((void**) &dev_gave, blocksPerGrid*sizeof(float) );
	gnumave = (int*) malloc(blocksPerGrid * sizeof(int));
	hipMalloc((void**) &dev_gnumave, blocksPerGrid*sizeof(int) );
    int allnumave = 0;

//if using regularization, calculate average value of voxels and substitute too large voxels value.
	ofstream fpriorout;
	float hostAvep;
    if(rgl == 1){
	allave = 0.0;
	calave<<<blocksPerGrid, reducsize>>>(dev_poimage, dev_gave);
	hipMemcpy(gave, dev_gave, blocksPerGrid*sizeof(float), hipMemcpyDeviceToHost);
	for(int jj=0; jj< blocksPerGrid; jj++)  allave += gave[jj];
	allave /= msize;
	hipMemcpyToSymbol(HIP_SYMBOL(avep), &allave, sizeof(float), 0, hipMemcpyHostToDevice);

	cout<<"Prior image average value, before filter: "<<allave<<endl;

	hipMemcpyToSymbol(HIP_SYMBOL(aveunf), &allave, sizeof(float), 0, hipMemcpyHostToDevice);
	allave = 0.0;
	allnumave = 0;
	calavewithfilter<<<blocksPerGrid, reducsize>>>(dev_poimage, dev_gave,dev_gnumave);
	gpuErrchk(hipPeekAtLastError());gpuErrchk(hipDeviceSynchronize());
	hipMemcpy(gave, dev_gave, blocksPerGrid*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(gnumave, dev_gnumave, blocksPerGrid*sizeof(int), hipMemcpyDeviceToHost);
	for(int jj=0; jj< blocksPerGrid; jj++)  {allave += gave[jj]; allnumave += gnumave[jj];}
	allave /= allnumave;
	hipMemcpyToSymbol(HIP_SYMBOL(avep), &allave, sizeof(float), 0, hipMemcpyHostToDevice);
	hostAvep = allave;	//for storing A_P

	cout<<"Prior image average value, after filter: "<<allave<<" "<<allnumave<<endl;
	hipMemcpy( poimage, dev_poimage, msize*sizeof(float), hipMemcpyDeviceToHost );
		fpriorout.open("priorImage", ios::out | ios::binary);
		for(int iii=0; iii< msize; iii++)
		{
			fpriorout.write( (char*)&poimage[iii], sizeof(float));
		}
		fpriorout.close();
    }

	vector<double> logLikelihood, logR;	//value of objective functions in all iterations
//start iterations for image reconstruction
	for(int ij=0; ij<itenum; ij++){

		double templogLikelihood = 0.0;
		double templogR = 0.0;

		cout<<"Starting "<<ij<<" iteration."<<endl;

//        if(rgl == 1)
//        {
//            //sumimp = 0.;
//            //for(int jj=0; jj<msize; jj++) sumimp += smatrix[jj];
//			if(blur == 1) 
//			{
//				gaussianblur(nx, ny, nz);
//				sumb = 0.;
//				for(int jj=0; jj<msize; jj++) sumb += bmatrix[jj];
//			}
//		}

		timinggpu.StartCounter();
		cout<<"pre1 "<<smatrix[0]<<" "<<smatrix[100]<<endl;
		cout<<"pre2 "<<snmatrix[0]<<" "<<snmatrix[100]<<endl;
		cout<<"pre3 "<<lastimage[0]<<" "<<lastimage[100]<<endl;
		hipMemcpy( dev_smatrix, smatrix, msize*sizeof(float), hipMemcpyHostToDevice );
		hipMemset( dev_snmatrix, 0, msize*sizeof(float));
		hipMemset( dev_xlor.linevalue, 0, nummainaxis[0]*sizeof(float));
		hipMemset( dev_ylor.linevalue, 0, nummainaxis[1]*sizeof(float)); 
		hipMemset( dev_zlor.linevalue, 0, nummainaxis[2]*sizeof(float)); 
        //hipMemset( dev_deri, 0.0f, msize*sizeof(float));
		timeall.memoryIO += timinggpu.GetCounter();


		timinggpu.StartCounter();
		xfpro<<<blocksPerGrid, threads>>>(dev_xlor, dev_smatrix);
		timeall.txforward += timinggpu.GetCounter();

		timinggpu.StartCounter();
		yfpro<<<blocksPerGrid, threads>>>(dev_ylor, dev_smatrix);
		timeall.tyforward += timinggpu.GetCounter();

		timinggpu.StartCounter();
		zfpro<<<blocksPerGrid, threads>>>(dev_zlor, dev_smatrix);
		timeall.tzforward += timinggpu.GetCounter();

		timinggpu.StartCounter();
		xbpro<<<blocksPerGrid, threads>>>(dev_xlor, dev_snmatrix);
		timeall.txbackward += timinggpu.GetCounter();

		timinggpu.StartCounter();
		ybpro<<<blocksPerGrid, threads>>>(dev_ylor, dev_snmatrix);
		timeall.tybackward += timinggpu.GetCounter();

		timinggpu.StartCounter();
		zbpro<<<blocksPerGrid, threads>>>(dev_zlor, dev_snmatrix);
        timeall.tzbackward += timinggpu.GetCounter();


	//	testf<<<blocksPerGrid, threads>>>(dev_test);
	//	testf<<<1,1>>>(dev_test);

	//	hipMemcpy(snmatrix, dev_snmatrix, msize*sizeof(float), hipMemcpyDeviceToHost);
	//	hipMemcpy(ylor.linevalue, dev_ylor.x1, nummainaxis[1]*sizeof(float), hipMemcpyDeviceToHost);
	//	cout<<ylor.linevalue[0]<<" "<<ylor.linevalue[1]<<" "<<ylor.linevalue[2]<<" "<<ylor.linevalue[3]<<" "<<ylor.linevalue[4]<<" "<<ylor.linevalue[5]<<endl;

	//	hipMemcpy(xlor.linevalue, dev_xlor.linevalue, nummainaxis[0]*sizeof(float), hipMemcpyDeviceToHost);
	//	string filein=argv[1];
	//	cout<<xlor.linevalue[0]<<" "<<xlor.linevalue[1]<<endl;
		
	//	hipMemcpy(testmatrix, dev_testmatrix, sharesize * sharesize *sizeof(float), hipMemcpyDeviceToHost);

	//	for(int ii=0; ii<sharesize; ii++)
	//	{
	//	for(int ij = 0; ij<sharesize; ij++)
	//	{cout<<testmatrix[ii*sharesize + ij]<<" ";}
	//	cout<<endl;
	//	}

	//	hipMemcpy(test, dev_test, sizeof(float), hipMemcpyDeviceToHost);
	//	cout<<"Test data is "<<test[0]<<endl;

		sumde2 = 0.0;

		//no regularization
		if(rgl == 0)
		{
			if(norma == 0){
				timinggpu.StartCounter();
				calnewmatrix000<<<blocksPerGrid, threads>>>(dev_snmatrix, dev_smatrix);//snmatrix[jj] = smatrix[jj] * snmatrix[jj];
				timeall.tpostimageprocess += timinggpu.GetCounter();
			}
			else{ 
				timinggpu.StartCounter();
				calnewmatrix100<<<blocksPerGrid, threads>>>(dev_snmatrix, dev_smatrix, dev_normimage);
				timeall.tpostimageprocess += timinggpu.GetCounter();
			}
		}
		
		//regularization, no blur
		else if(rgl == 1 && blur == 0)
		{
            //calculate average value without filter
			//allave = 0.0;
			//calave<<<blocksPerGrid, reducsize>>>(dev_smatrix, dev_gave);
			//hipMemcpy(gave, dev_gave, blocksPerGrid*sizeof(float), hipMemcpyDeviceToHost);
			//for(int jj=0; jj< blocksPerGrid; jj++)  allave += gave[jj];
			//allave /= msize;
			//hipMemcpyToSymbol(HIP_SYMBOL(aves), &allave, sizeof(float), 0, hipMemcpyHostToDevice);

            ////calculate average value with filter
			//hipMemcpyToSymbol(HIP_SYMBOL(aveunf), &allave, sizeof(float), 0, hipMemcpyHostToDevice);
			//allave = 0.0;
			//allnumave = 0;
			//calavewithfilter<<<blocksPerGrid, reducsize>>>(dev_smatrix, dev_gave,dev_gnumave);
			//hipMemcpy(gave, dev_gave, blocksPerGrid*sizeof(float), hipMemcpyDeviceToHost);
			//hipMemcpy(gnumave, dev_gnumave, blocksPerGrid*sizeof(int), hipMemcpyDeviceToHost);
			//for(int jj=0; jj< blocksPerGrid; jj++)  {allave += gave[jj]; allnumave += gnumave[jj];}
			//allave /= allnumave;
			//hipMemcpyToSymbol(HIP_SYMBOL(aves), &allave, sizeof(float), 0, hipMemcpyHostToDevice);



			if(norma == 0){
				timinggpu.StartCounter();
				calnewmatrix010<<<blocksPerGrid, threads>>>(dev_snmatrix, dev_smatrix, dev_poimage);//Error check: gpuErrchk(hipPeekAtLastError());gpuErrchk(hipDeviceSynchronize());}
				timeall.tpostimageprocess += timinggpu.GetCounter();
			}
			else{ 
				timinggpu.StartCounter();
				calnewmatrix110<<<blocksPerGrid, threads>>>(dev_snmatrix, dev_smatrix, dev_normimage, dev_poimage);
				timeall.tpostimageprocess += timinggpu.GetCounter();
			}
		}

		//regularizatin, blur
		else if(rgl == 1 && blur == 1)
		{
			hipMemset( dev_bmatrix, 0, msize*sizeof(float));
			hipMemset( dev_allweight, 0, msize*sizeof(float));
			gpublur<<<blocksPerGrid, threads>>>(dev_smatrix, dev_bmatrix, dev_allweight);	//blur image from last iteration

            //calculate average value for the blurred image without filter
			//allave = 0.0;
			//calave<<<blocksPerGrid, reducsize>>>(dev_bmatrix, dev_gave);
			//hipMemcpy(gave, dev_gave, blocksPerGrid*sizeof(float), hipMemcpyDeviceToHost);
			//for(int jj=0; jj< blocksPerGrid; jj++)  allave += gave[jj];
			//allave /= msize;
			//hipMemcpyToSymbol(HIP_SYMBOL(aves), &allave, sizeof(float), 0, hipMemcpyHostToDevice);

            //calculate average value with filter
			//hipMemcpyToSymbol(HIP_SYMBOL(aveunf), &allave, sizeof(float), 0, hipMemcpyHostToDevice);
			//allave = 0.0;
			//allnumave = 0;
			//calavewithfilter<<<blocksPerGrid, reducsize>>>(dev_bmatrix, dev_gave,dev_gnumave);
			//hipMemcpy(gave, dev_gave, blocksPerGrid*sizeof(float), hipMemcpyDeviceToHost);
			//hipMemcpy(gnumave, dev_gnumave, blocksPerGrid*sizeof(int), hipMemcpyDeviceToHost);
			//for(int jj=0; jj< blocksPerGrid; jj++)  {allave += gave[jj]; allnumave += gnumave[jj];}
			//allave /= allnumave;
			//hipMemcpyToSymbol(HIP_SYMBOL(aves), &allave, sizeof(float), 0, hipMemcpyHostToDevice);

			//calculate new image for this iteration
			if(norma == 0){
				timinggpu.StartCounter();
				calnewmatrix011<<<blocksPerGrid, threads>>>(dev_snmatrix, dev_smatrix, dev_poimage, dev_bmatrix, dev_allweight);//Error check: gpuErrchk(hipPeekAtLastError());gpuErrchk(hipDeviceSynchronize());}
				timeall.tpostimageprocess += timinggpu.GetCounter();
			}
			else{
				timinggpu.StartCounter();
				calnewmatrix111<<<blocksPerGrid, threads>>>(dev_snmatrix, dev_smatrix, dev_normimage, dev_poimage, dev_bmatrix, dev_allweight);
				timeall.tpostimageprocess += timinggpu.GetCounter();
			}

		}

		else cout<<"Unknown indentifier for regularization or blur!!!"<<endl;

		timinggpu.StartCounter();
		hipMemcpy(snmatrix, dev_snmatrix, msize*sizeof(float), hipMemcpyDeviceToHost);
		timeall.memoryIO += timinggpu.GetCounter();

		cout<<"pre2 "<<snmatrix[0]<<" "<<snmatrix[100]<<endl;
		cout<<"Finish "<<ij<<" iteration."<<endl;

//write new image to file.
		ostringstream convert;
		convert<<(ij+1);
		ofstream fout;
		string fileout=argv[2];
		fileout.append(convert.str());
		fout.open(fileout.c_str(), ios::out | ios::binary);
		if (fout.is_open()){
			for(int iii=0; iii< msize; iii++)
			{
				fout.write( (char*)&snmatrix[iii], sizeof(float));
			}
		}
		else cout<<"Unable to write image to file!!"<<endl;

		fout.close();

		//calculate objective function values
		double *gloglike, *dev_gloglike;
		gloglike = (double*) malloc(blocksPerGrid * sizeof(double));
		hipMalloc((void**) &dev_gloglike, blocksPerGrid*sizeof(double) );
		calLogLike<<<blocksPerGrid, reducsize>>>(dev_xlor.linevalue, dev_gloglike, nummainaxis[0]);
		hipMemcpy(gloglike, dev_gloglike, blocksPerGrid*sizeof(double), hipMemcpyDeviceToHost);
		for(int iobj = 0; iobj < blocksPerGrid; iobj++) templogLikelihood += gloglike[iobj];
	
		calLogLike<<<blocksPerGrid, reducsize>>>(dev_ylor.linevalue, dev_gloglike, nummainaxis[1]);
		hipMemcpy(gloglike, dev_gloglike, blocksPerGrid*sizeof(double), hipMemcpyDeviceToHost);
		for(int iobj = 0; iobj < blocksPerGrid; iobj++) templogLikelihood += gloglike[iobj];
	
		calLogLike<<<blocksPerGrid, reducsize>>>(dev_zlor.linevalue, dev_gloglike, nummainaxis[2]);
		hipMemcpy(gloglike, dev_gloglike, blocksPerGrid*sizeof(double), hipMemcpyDeviceToHost);
		for(int iobj = 0; iobj < blocksPerGrid; iobj++) templogLikelihood += gloglike[iobj];


		calLogLikeS<<<blocksPerGrid, reducsize>>>(dev_smatrix, dev_normimage, dev_gloglike, msize, norma);
		hipMemcpy(gloglike, dev_gloglike, blocksPerGrid*sizeof(double), hipMemcpyDeviceToHost);
		for(int iobj = 0; iobj < blocksPerGrid; iobj++) templogLikelihood += gloglike[iobj];


		if(rgl == 1 && blur == 0) calLogR<<<blocksPerGrid, reducsize>>>(dev_smatrix, dev_poimage, dev_gloglike, msize);
		else if(rgl == 1 && blur == 1) calLogR<<<blocksPerGrid, reducsize>>>(dev_bmatrix, dev_poimage, dev_gloglike, msize);
		if(rgl == 1) {
			hipMemcpy(gloglike, dev_gloglike, blocksPerGrid*sizeof(double), hipMemcpyDeviceToHost);
			for(int iobj = 0; iobj < blocksPerGrid; iobj++) templogR += gloglike[iobj];	
		}

		//hipMemcpy(xlor.linevalue, dev_xlor.linevalue, nummainaxis[0]*sizeof(float), hipMemcpyDeviceToHost);
		//hipMemcpy(ylor.linevalue, dev_ylor.linevalue, nummainaxis[1]*sizeof(float), hipMemcpyDeviceToHost);
		//hipMemcpy(zlor.linevalue, dev_zlor.linevalue, nummainaxis[2]*sizeof(float), hipMemcpyDeviceToHost);
		//for(int iobj = 0; iobj < nummainaxis[0]; iobj++) {
		//	if (xlor.linevalue[iobj] > ThreshLineValue ) templogLikelihood += log(xlor.linevalue[iobj]);
		//}
		//for(int iobj = 0; iobj < nummainaxis[1]; iobj++) {
		//	if (ylor.linevalue[iobj] > ThreshLineValue) templogLikelihood += log(ylor.linevalue[iobj]);
		//}
		//for(int iobj = 0; iobj < nummainaxis[2]; iobj++) {
		//	if (zlor.linevalue[iobj] > ThreshLineValue) templogLikelihood += log(zlor.linevalue[iobj]);
		//}

		//for(int iobj = 0; iobj < msize; iobj++){
		//	if(norm == 0) templogLikelihood -= smatrix[iobj];
		//	else {
		//		float tempSen = normimage[iobj] / maxnorm;
		//		if(tempSen < ThreshNorm) tempSen = ThreshNorm;
		//		templogLikelihood -= tempSen * smatrix[iobj];
		//	}
		//}
		//
		//if(rgl == 1 && blur == 0) for(int iobj = 0; iobj < msize; iobj++) templogR -= pow(smatrix[iobj]/hostAve - poimage[iobj]/hostAvep,2);
		//else if(rgl == 1 && blur == 1) {
		//	hipMemcpy(bmatrix, dev_bmatrix, msize * sizeof(float), hipMemcpyDeviceToHost);
		//	for(int iobj = 0; iobj < msize; iobj++) templogR -= pow(bmatrix[iobj]/hostAve - poimage[iobj]/hostAvep,2);
		//}

		templogR *= beta;
		
		logLikelihood.push_back(templogLikelihood);
		logR.push_back(templogR);

		//prepare for next iteration
		for(int iii=0; iii< msize; iii++)
		{
			smatrix[iii] = snmatrix[iii];
			snmatrix[iii] = 0.;
		}


	}

	ofstream fObjFunc ("ObjectiveFuncValue.txt");
	if(fObjFunc.is_open()){
		for (int i=0; i< itenum; i++) fObjFunc << i << " "<< logLikelihood[i] << " " << logR[i] << " " << logLikelihood[i] + logR[i] << endl;
	}
	else cout<< "Can not open ObjectiveFuncValue.txt!!" <<endl;
	fObjFunc.close();

	timeall.printvalue();	//print out timing information about cuda execution.

	//cout<< "Total number of voxel lor distance calculation is "<< numcal << endl;
	//cout<< "Total number of voxel lor kernel calculation is " << numcal2 << endl;
	hipFree(dev_xlor.x1);
	hipFree(dev_xlor.y1);
	hipFree(dev_xlor.z1);
	hipFree(dev_xlor.x2);
	hipFree(dev_xlor.y2);
	hipFree(dev_xlor.z2);
    hipFree(dev_ylor.x1);
    hipFree(dev_ylor.y1);
    hipFree(dev_ylor.z1);
    hipFree(dev_ylor.x2);
    hipFree(dev_ylor.y2);
    hipFree(dev_ylor.z2);
    hipFree(dev_zlor.x1);
    hipFree(dev_zlor.y1);
    hipFree(dev_zlor.z1);
    hipFree(dev_zlor.x2);
    hipFree(dev_zlor.y2);
    hipFree(dev_zlor.z2);
	hipFree(dev_xlor.linevalue);
	hipFree(dev_ylor.linevalue);
	hipFree(dev_zlor.linevalue);
	hipFree(dev_smatrix);
	hipFree(dev_snmatrix);
	hipFree(dev_poimage);
	hipFree(dev_deri);
	hipFree(dev_bmatrix);
    free(xlor.x1);
    free(xlor.y1);
    free(xlor.z1);
    free(xlor.x2);
    free(xlor.y2);
    free(xlor.z2);
    free(ylor.x1);
    free(ylor.y1);
    free(ylor.z1);
    free(ylor.x2);
    free(ylor.y2);
    free(ylor.z2);
    free(zlor.x1);
    free(zlor.y1);
    free(zlor.z1);
    free(zlor.x2);
    free(zlor.y2);
    free(zlor.z2);	
	free(xlor.linevalue);
	free(ylor.linevalue);
	free(zlor.linevalue);
	free(smatrix);
	free(snmatrix);
	free(bmatrix);
	free(poimage);
	free(deri);
	free(allweight);
	return 0;
}

vector<string> explode(string s, char c)
{
	string buff="";
	vector<string> v;
	char n;
	
	for(unsigned i=0; i<s.length(); ++i)
	{
		n=s.at(i);
		if(n != c) buff+=n; else
		if(n == c && buff != "") { v.push_back(buff); buff = ""; }
	}
	if(buff != "") v.push_back(buff);
	
	return v;
}



int gaussianblur(int nx, int ny, int nz)
{
    float sumweight;

	int ci,cj,ck,li,hi,lj,hj,lk,hk;
	for(int i=0; i< msize;i++ )
	{
		bmatrix[i] = 0.;
		ci = (i % (nx*ny)) % nx;
		cj = (i % (nx*ny)) / nx;
		ck = i / (nx*ny);
		li = max(0, ci - indr[0]);
		lj = max(0, cj - indr[1]);
		lk = max(0, ck - indr[2]);
		hi = min(nx - 1, ci + indr[0]);
		hj = min(ny - 1, cj + indr[1]);
		hk = min(nz - 1, ck + indr[2]);
        sumweight = 0.0;

		for(int ii= li; ii<= hi; ii++)
		{
			for(int jj = lj; jj<= hj; jj++)
			{
				for(int kk = lk; kk<= hk; kk++)
				{
					if((pow((ii-ci)/rads[0],2) + pow((jj-cj)/rads[1],2) + pow((kk-ck)/rads[2],2)) * pow(a,2) < 1.0) 
					{
						bmatrix[i] += smatrix[ii + jj * nx + kk * nx * ny] * exp(-(pow(ii-ci,2)/(2. * pow(bsgm[0],2)) + pow(jj-cj,2)/(2. * pow(bsgm[1],2)) + pow(kk-ck,2)/(2. * pow(bsgm[2],2))) * pow(a,2));
						sumweight += exp(-(pow(ii-ci,2)/(2. * pow(bsgm[0],2)) + pow(jj-cj,2)/(2. * pow(bsgm[1],2)) + pow(kk-ck,2)/(2. * pow(bsgm[2],2))) * pow(a,2));
					}

				}
			}
		}
		allweight[i] = sumweight;
		bmatrix[i] = bmatrix[i] / sumweight;
		//for(int j=li; j<=hi; j++) bmatrix[i] += smatrix[j + cj * nx + ck * nx * ny] * exp(-pow(j-ci,2) * pow(a,2)/(2. * pow(bsgm,2)));
	}
	return 0;
}

//function that read lor from fin, sort lor, and copy lor to cuda
int preplor(string filein, int senmap)
{
	ifstream fin;
	fin.open(filein.c_str(), ios::in | ios::binary);

	vector<lor> alllor;		//matrix for all lor
	numline = 0;
	string line;
	
	nummainaxis[0] = 0;
	nummainaxis[1] = 0;
	nummainaxis[2] = 0;

	timing.StartCounter();
	if (fin.is_open()){

		while ( !fin.eof() )
		{
			numline += 1;
			float coordlor[6];
			fin.read((char*)coordlor, 6 * sizeof(float));
			lor bufflor;
			if(wgt == 1 && senmap == 0) {
				float aa,bb,cc, delta, ts1, ts2, ulen2;
				aa = pow(coordlor[0]-coordlor[3],2) + pow(coordlor[1]-coordlor[4],2);
				bb = 2. * (coordlor[0] * (coordlor[3] - coordlor[0]) + coordlor[1] * (coordlor[4] - coordlor[1]));
				cc = pow(coordlor[0], 2) + pow(coordlor[1], 2) - pow(CylRadius,2);
				ulen2 = pow(coordlor[0]-coordlor[3],2) + pow(coordlor[1]-coordlor[4],2) + pow(coordlor[2]-coordlor[5],2);
				delta = pow(bb,2) - 4.* aa * cc;
		
				if(delta <= 0.) bufflor.weight = 1.0;
				else
				{
					ts1 = (-bb + sqrt(delta)) / (2. * aa);
					ts2 = (-bb - sqrt(delta)) / (2. * aa);
					if(coordlor[5] == coordlor[2] && (coordlor[2] > CylHeight / 2.0 || coordlor[2] < -CylHeight / 2.0)) ts1 = ts2;
					else {
						float zz1 = coordlor[2] + ts1 * (coordlor[5] - coordlor[2]);
						if(zz1 > CylHeight / 2.0) ts1 = (CylHeight/2.0 - coordlor[2]) / (coordlor[5] - coordlor[2]);
						else if(zz1 < -CylHeight/2.0) ts1 = (-CylHeight/2.0 - coordlor[2]) / (coordlor[5] - coordlor[2]);
						float zz2 = coordlor[2] + ts2 * (coordlor[5] - coordlor[2]);
						if(zz2 > CylHeight / 2.0) ts2 = (CylHeight/2.0 - coordlor[2]) / (coordlor[5] - coordlor[2]);
						else if(zz2 < -CylHeight/2.0) ts2 = (-CylHeight/2.0 - coordlor[2]) / (coordlor[5] - coordlor[2]);
					}
			
					bufflor.weight = sqrt(ulen2) * abs(ts1 - ts2);
					bufflor.weight = exp(-(MU * bufflor.weight));

				}

			}
			else bufflor.weight = 1.0;
			bufflor.x1 = coordlor[0] - (-bndry[0]/2. + 0.5 * a);
			bufflor.y1 = coordlor[1] - (-bndry[1]/2. + 0.5 * a);
			bufflor.z1 = coordlor[2] - (-bndry[2]/2. + 0.5 * a);
			bufflor.x2 = coordlor[3] - (-bndry[0]/2. + 0.5 * a);
			bufflor.y2 = coordlor[4] - (-bndry[1]/2. + 0.5 * a);
			bufflor.z2 = coordlor[5] - (-bndry[2]/2. + 0.5 * a);
	
			if(abs(bufflor.x1-bufflor.x2) >= abs(bufflor.y1-bufflor.y2) && abs(bufflor.x1-bufflor.x2) >= abs(bufflor.z1-bufflor.z2)) {bufflor.mainaxis = 0; nummainaxis[0] += 1;}
			else if(abs(bufflor.y1-bufflor.y2) >= abs(bufflor.x1-bufflor.x2) && abs(bufflor.y1-bufflor.y2) >= abs(bufflor.z1-bufflor.z2)) {bufflor.mainaxis = 1; nummainaxis[1] += 1;}
			else if(abs(bufflor.z1-bufflor.z2) >= abs(bufflor.x1-bufflor.x2) && abs(bufflor.z1-bufflor.z2) >= abs(bufflor.y1-bufflor.y2)) {bufflor.mainaxis = 2; nummainaxis[2] += 1;}
			else cout<<"Cannot fing the main axis!!"<<endl;
	
			alllor.push_back(bufflor);
		}
	}
	else cout<<"Unable to open input lor file!!"<<endl;

	fin.close();
	timeall.lorsorting += timing.GetCounter();
//	cout<<"before lor malloc"<<endl;

	timing.StartCounter();
	xlor.x1 = (float*) malloc(nummainaxis[0] * sizeof(float)); 
	xlor.y1 = (float*) malloc(nummainaxis[0] * sizeof(float));
	xlor.z1 = (float*) malloc(nummainaxis[0] * sizeof(float));
	xlor.x2 = (float*) malloc(nummainaxis[0] * sizeof(float));
	xlor.y2 = (float*) malloc(nummainaxis[0] * sizeof(float));
	xlor.z2 = (float*) malloc(nummainaxis[0] * sizeof(float));
	xlor.linevalue = (float*) malloc(nummainaxis[0] * sizeof(float));
	xlor.weight = (float*) malloc(nummainaxis[0] * sizeof(float));

    ylor.x1 = (float*) malloc(nummainaxis[1] * sizeof(float)); 
    ylor.y1 = (float*) malloc(nummainaxis[1] * sizeof(float));
    ylor.z1 = (float*) malloc(nummainaxis[1] * sizeof(float));
    ylor.x2 = (float*) malloc(nummainaxis[1] * sizeof(float));
    ylor.y2 = (float*) malloc(nummainaxis[1] * sizeof(float));
    ylor.z2 = (float*) malloc(nummainaxis[1] * sizeof(float));
	ylor.linevalue = (float*) malloc(nummainaxis[1] * sizeof(float));
	ylor.weight = (float*) malloc(nummainaxis[1] * sizeof(float));

    zlor.x1 = (float*) malloc(nummainaxis[2] * sizeof(float)); 
    zlor.y1 = (float*) malloc(nummainaxis[2] * sizeof(float));
    zlor.z1 = (float*) malloc(nummainaxis[2] * sizeof(float));
    zlor.x2 = (float*) malloc(nummainaxis[2] * sizeof(float));
    zlor.y2 = (float*) malloc(nummainaxis[2] * sizeof(float));
    zlor.z2 = (float*) malloc(nummainaxis[2] * sizeof(float));
	zlor.linevalue = (float*) malloc(nummainaxis[2] * sizeof(float));
	zlor.weight = (float*) malloc(nummainaxis[2] * sizeof(float));
	timeall.memoryIO += timing.GetCounter();

//	cout<<"after lor malloc"<<endl;
	timing.StartCounter();
	int cma[3] = {0,0,0};	//structure of arrays. 
	for(int i=0; i< numline; i++)
	{
		lor bufflor = alllor[i];
		if(bufflor.mainaxis == 0) 
		{
			xlor.x1[cma[0]] = bufflor.x1;
			xlor.y1[cma[0]] = bufflor.y1;
			xlor.z1[cma[0]] = bufflor.z1;
			xlor.x2[cma[0]] = bufflor.x2;
			xlor.y2[cma[0]] = bufflor.y2;
			xlor.z2[cma[0]] = bufflor.z2;
			xlor.weight[cma[0]] = bufflor.weight;
			cma[0] += 1;
		}
		else if(bufflor.mainaxis == 1)
		{
            ylor.x1[cma[1]] = bufflor.x1;
            ylor.y1[cma[1]] = bufflor.y1;
            ylor.z1[cma[1]] = bufflor.z1;
            ylor.x2[cma[1]] = bufflor.x2;
            ylor.y2[cma[1]] = bufflor.y2;
            ylor.z2[cma[1]] = bufflor.z2;
			ylor.weight[cma[1]] = bufflor.weight;

            cma[1] += 1;
	
		}
		else if(bufflor.mainaxis == 2)
		{
            zlor.x1[cma[2]] = bufflor.x1;
            zlor.y1[cma[2]] = bufflor.y1;
            zlor.z1[cma[2]] = bufflor.z1;
            zlor.x2[cma[2]] = bufflor.x2;
            zlor.y2[cma[2]] = bufflor.y2;
            zlor.z2[cma[2]] = bufflor.z2;
			zlor.weight[cma[2]] = bufflor.weight;
            cma[2] += 1;
		}
	}
	if(cma[0] !=  nummainaxis[0] || cma[1] != nummainaxis[1] || cma[2] != nummainaxis[2]) cout<< "Something wrong with the number of lors for each main axis!!" <<endl;


	vector<lor>().swap(alllor);		//deallocate lor

	timeall.lorsorting += timing.GetCounter();

//	icout<<"before cudamalloc"<<endl;
	timinggpu.StartCounter();
    hipMalloc((void**) &dev_xlor.x1, nummainaxis[0]*sizeof(float) );
	hipMalloc((void**) &dev_xlor.y1, nummainaxis[0]*sizeof(float) ); 
	hipMalloc((void**) &dev_xlor.z1, nummainaxis[0]*sizeof(float) ); 
	hipMalloc((void**) &dev_xlor.x2, nummainaxis[0]*sizeof(float) ); 
	hipMalloc((void**) &dev_xlor.y2, nummainaxis[0]*sizeof(float) ); 
	hipMalloc((void**) &dev_xlor.z2, nummainaxis[0]*sizeof(float) ); 	
	hipMalloc((void**) &dev_xlor.linevalue, nummainaxis[0]*sizeof(float) );
	hipMalloc((void**) &dev_xlor.weight, nummainaxis[0]*sizeof(float) );

    hipMalloc((void**) &dev_ylor.x1, nummainaxis[1]*sizeof(float) );
    hipMalloc((void**) &dev_ylor.y1, nummainaxis[1]*sizeof(float) ); 
    hipMalloc((void**) &dev_ylor.z1, nummainaxis[1]*sizeof(float) ); 
    hipMalloc((void**) &dev_ylor.x2, nummainaxis[1]*sizeof(float) ); 
    hipMalloc((void**) &dev_ylor.y2, nummainaxis[1]*sizeof(float) ); 
    hipMalloc((void**) &dev_ylor.z2, nummainaxis[1]*sizeof(float) );
    hipMalloc((void**) &dev_ylor.linevalue, nummainaxis[1]*sizeof(float) );
	hipMalloc((void**) &dev_ylor.weight, nummainaxis[1]*sizeof(float) );	

    hipMalloc((void**) &dev_zlor.x1, nummainaxis[2]*sizeof(float) );
    hipMalloc((void**) &dev_zlor.y1, nummainaxis[2]*sizeof(float) ); 
    hipMalloc((void**) &dev_zlor.z1, nummainaxis[2]*sizeof(float) ); 
    hipMalloc((void**) &dev_zlor.x2, nummainaxis[2]*sizeof(float) ); 
    hipMalloc((void**) &dev_zlor.y2, nummainaxis[2]*sizeof(float) ); 
    hipMalloc((void**) &dev_zlor.z2, nummainaxis[2]*sizeof(float) ); 
	hipMalloc((void**) &dev_zlor.linevalue, nummainaxis[2]*sizeof(float) );
	hipMalloc((void**) &dev_zlor.weight, nummainaxis[2]*sizeof(float) );

//	cout<<"before cuda memory copy"<<endl;
	hipMemcpy( dev_xlor.x1, xlor.x1, nummainaxis[0]*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( dev_xlor.y1, xlor.y1, nummainaxis[0]*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( dev_xlor.z1, xlor.z1, nummainaxis[0]*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( dev_xlor.x2, xlor.x2, nummainaxis[0]*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( dev_xlor.y2, xlor.y2, nummainaxis[0]*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( dev_xlor.z2, xlor.z2, nummainaxis[0]*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( dev_xlor.weight, xlor.weight, nummainaxis[0]*sizeof(float), hipMemcpyHostToDevice );

    hipMemcpy( dev_ylor.x1, ylor.x1, nummainaxis[1]*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dev_ylor.y1, ylor.y1, nummainaxis[1]*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dev_ylor.z1, ylor.z1, nummainaxis[1]*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dev_ylor.x2, ylor.x2, nummainaxis[1]*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dev_ylor.y2, ylor.y2, nummainaxis[1]*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dev_ylor.z2, ylor.z2, nummainaxis[1]*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( dev_ylor.weight, ylor.weight, nummainaxis[1]*sizeof(float), hipMemcpyHostToDevice );

    hipMemcpy( dev_zlor.x1, zlor.x1, nummainaxis[2]*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dev_zlor.y1, zlor.y1, nummainaxis[2]*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dev_zlor.z1, zlor.z1, nummainaxis[2]*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dev_zlor.x2, zlor.x2, nummainaxis[2]*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dev_zlor.y2, zlor.y2, nummainaxis[2]*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dev_zlor.z2, zlor.z2, nummainaxis[2]*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( dev_zlor.weight, zlor.weight, nummainaxis[2]*sizeof(float), hipMemcpyHostToDevice );
	timeall.memoryIO += timinggpu.GetCounter();

    return 0;
}

//calculate snmatrix based on projection and previous value. No normalization, no regularization, no blur
__global__ void calnewmatrix000(float *snmatrix, float *smatrix)
{
	int nx = d_imageindex[0], ny = d_imageindex[1], nz = d_imageindex[2];
	int x = threadIdx.x, y = threadIdx.y, z = blockIdx.x;
	int jj;	//image index in 1D
	while(z < nz)
	{
		y = threadIdx.y;
		while(y < ny)
		{
			x = threadIdx.x;
			while(x < nx)
			{
			    jj = x + y * nx + z * nx * ny;
				snmatrix[jj] = snmatrix[jj] * smatrix[jj];
				x += blockDim.x;
			}
			y += blockDim.y;
		}
		z += gridDim.x;
	}
}

//Yes normalization, no regularization, no blur
__global__ void calnewmatrix100(float *snmatrix, float *smatrix, float *normimage)
{
    int nx = d_imageindex[0], ny = d_imageindex[1], nz = d_imageindex[2];
    int x = threadIdx.x, y = threadIdx.y, z = blockIdx.x;
	int jj;
	float senratio, maxnorm = d_norm[0], ThreshNorm = d_norm[1];
	while(z < nz)
	{
		y = threadIdx.y;
		while(y < ny)
		{
			x = threadIdx.x;
			while(x < nx)
			{
			    jj = x + y * nx + z * nx * ny;
				if(normimage[jj] / maxnorm < ThreshNorm) senratio = ThreshNorm;
				else senratio = normimage[jj] / maxnorm;
       			snmatrix[jj] = snmatrix[jj] * smatrix[jj] / senratio;
				x += blockDim.x;
			}
			y += blockDim.y;
		}
		z += gridDim.x;
	}
}       


//No normalization, yes regularization, no blur
__global__ void calnewmatrix010(float *snmatrix, float *smatrix, float *poimage)
{
	int nx = d_imageindex[0], ny = d_imageindex[1], nz = d_imageindex[2];
    int x = threadIdx.x, y = threadIdx.y, z = blockIdx.x;
    int jj; //image index in 1D
	float beta = d_info[3], aa, bb, cc, laves = aves[0], lavep = avep[0];
    while(z < nz)
    {
        y = threadIdx.y;
        while(y < ny)
        {
            x = threadIdx.x;
            while(x < nx)
            {
                jj = x + y * nx + z * nx * ny;
				aa = 2.0f * beta / powf(laves,2);
				bb = 1.0f - 2.0f * beta * poimage[jj] / (laves * lavep);
				cc = -snmatrix[jj] * smatrix[jj];
                snmatrix[jj] = (-bb + sqrtf(powf(bb,2) - 4.0f * aa * cc)) / (2.0f * aa);
                x += blockDim.x;
            }
            y += blockDim.y;
        }
        z += gridDim.x;
    }

}

//Yes normalization, yes regularization, no blur
__global__ void calnewmatrix110(float *snmatrix, float *smatrix, float *normimage, float *poimage)
{
	int nx = d_imageindex[0], ny = d_imageindex[1], nz = d_imageindex[2];
    int x = threadIdx.x, y = threadIdx.y, z = blockIdx.x;
    int jj; //image index in 1D
	float beta = d_info[3], aa, bb, cc, laves = aves[0], lavep = avep[0];
	float senratio, maxnorm = d_norm[0], ThreshNorm = d_norm[1];
    while(z < nz)
    {
        y = threadIdx.y;
        while(y < ny)
        {
            x = threadIdx.x;
            while(x < nx)
            {
                jj = x + y * nx + z * nx * ny;
				if(normimage[jj] / maxnorm < ThreshNorm) senratio = ThreshNorm;
				else senratio = normimage[jj] / maxnorm;
				aa = 2.0f * beta / powf(laves,2);
				bb = senratio - 2.0f * beta * poimage[jj] / (laves * lavep);
				cc = -snmatrix[jj] * smatrix[jj];
                snmatrix[jj] = (-bb + sqrtf(powf(bb,2) - 4.0f * aa * cc)) / (2.0f * aa);
                x += blockDim.x;
            }
            y += blockDim.y;
        }
        z += gridDim.x;
    }

}


//No normalization, ues regularization, yes blur.
__global__ void calnewmatrix011(float *snmatrix, float *smatrix, float *poimage, float *bmatrix, float *allweight)
{
	int nx = d_imageindex[0], ny = d_imageindex[1], nz = d_imageindex[2];
    int x = threadIdx.x, y = threadIdx.y, z = blockIdx.x;
    int jjj; //image index in 1D
	float beta = d_info[3], aa, bb, cc, laves = aves[0], lavep = avep[0], wi, a=d_info[0];
	int li,hi,lj,hj,lk,hk, idxy;
    while(z < nz)
    {
        y = threadIdx.y;
        while(y < ny)
        {
            x = threadIdx.x;
            while(x < nx)
            {
                jjj = x + y * nx + z * nx * ny;

				li = max(0, x - d_indr[0]);
				lj = max(0, y - d_indr[1]);
				lk = max(0, z - d_indr[2]);
				hi = min(nx - 1, x + d_indr[0]);
				hj = min(ny - 1, y + d_indr[1]);
				hk = min(nz - 1, z + d_indr[2]);

				aa = 0.0f;
				bb = 0.0f;
		
				for(int ii= li; ii<= hi; ii++)
				{
					for(int jj = lj; jj<= hj; jj++)
					{
						for(int kk = lk; kk<= hk; kk++)
						{
							if((powf((ii-x)/d_rads[0],2) + powf((jj-y)/d_rads[1],2) + powf((kk-z)/d_rads[2],2)) * powf(a,2) < 1.0) 
							{
								idxy = ii + jj * nx + kk * nx * ny;
                                wi = expf(-(powf(ii-x,2)/(2.0f * powf(d_bsgm[0],2)) + powf(jj-y,2)/(2.0f * powf(d_bsgm[1],2)) + powf(kk-z,2)/(2.0f * powf(d_bsgm[2],2))) * powf(a,2)) / allweight[idxy];
                                aa += wi;
                                bb += ((bmatrix[idxy] - smatrix[jjj] )/laves - poimage[idxy]/lavep) * wi;

							}
		
						}
					}
				}

				aa = aa * 2.0f * beta / powf(laves,2);
				bb = bb * 2.0f * beta / laves + 1.0f;
				cc = -snmatrix[jjj] * smatrix[jjj];
                snmatrix[jjj] = (-bb + sqrtf(powf(bb,2) - 4.0f * aa * cc)) / (2.0f * aa);
                x += blockDim.x;
            }
            y += blockDim.y;
        }
        z += gridDim.x;
    }

}

//Yes normalization, yes regularization, yes blur
__global__ void calnewmatrix111(float *snmatrix, float *smatrix, float *normimage, float *poimage, float *bmatrix, float *allweight)
{
	int nx = d_imageindex[0], ny = d_imageindex[1], nz = d_imageindex[2];
    int x = threadIdx.x, y = threadIdx.y, z = blockIdx.x;
    int jjj; //image index in 1D
	float beta = d_info[3], aa, bb, cc, laves = aves[0], lavep = avep[0], wi, a = d_info[0];
	int li,hi,lj,hj,lk,hk, idxy;
	float senratio, maxnorm = d_norm[0], ThreshNorm = d_norm[1];
    while(z < nz)
    {
        y = threadIdx.y;
        while(y < ny)
        {
            x = threadIdx.x;
            while(x < nx)
            {
                jjj = x + y * nx + z * nx * ny;
				if(normimage[jjj] / maxnorm < ThreshNorm) senratio = ThreshNorm;
				else senratio = normimage[jjj] / maxnorm;

				li = max(0, x - d_indr[0]);
				lj = max(0, y - d_indr[1]);
				lk = max(0, z - d_indr[2]);
				hi = min(nx - 1, x + d_indr[0]);
				hj = min(ny - 1, y + d_indr[1]);
				hk = min(nz - 1, z + d_indr[2]);

				aa = 0.0f;
				bb = 0.0f;
		
				for(int ii= li; ii<= hi; ii++)
				{
					for(int jj = lj; jj<= hj; jj++)
					{
						for(int kk = lk; kk<= hk; kk++)
						{
							if((powf((ii-x)/d_rads[0],2) + powf((jj-y)/d_rads[1],2) + powf((kk-z)/d_rads[2],2)) * powf(a,2) < 1.0) 
							{
								idxy = ii + jj * nx + kk * nx * ny;
                                wi = expf(-(powf(ii-x,2)/(2.0f * powf(d_bsgm[0],2)) + powf(jj-y,2)/(2.0f * powf(d_bsgm[1],2)) + powf(kk-z,2)/(2.0f * powf(d_bsgm[2],2))) * powf(a,2)) / allweight[idxy];
                                aa += wi;
                                bb += ((bmatrix[idxy] - smatrix[jjj])/laves - poimage[idxy]/lavep) * wi;

							}
		
						}
					}
				}

				aa = aa * 2.0f * beta / powf(laves,2);
				bb = bb * 2.0f * beta / laves + senratio;
				cc = -snmatrix[jjj] * smatrix[jjj];
                snmatrix[jjj] = (-bb + sqrtf(powf(bb,2) - 4.0f * aa * cc)) / (2.0f * aa);
                x += blockDim.x;
            }
            y += blockDim.y;
        }
        z += gridDim.x;
    }

}

//calculate average of voxel values
__global__ void calave(float *smatrix, float *gave)
{
	int msize = d_imageindex[3];
	int cacheindex = threadIdx.x, tid = threadIdx.x + blockIdx.x * blockDim.x;
   	__shared__ float buffave[reducsize];

	float buff = 0.0f;
	while(tid < msize)
	{
		buff += smatrix[tid];
		tid += blockDim.x * gridDim.x ;
	}
	buffave[cacheindex] = buff;
	__syncthreads();

	int i = blockDim.x / 2;
	while( i != 0)
	{
		if(cacheindex < i)  buffave[cacheindex] += buffave[cacheindex + i];
		__syncthreads();
		i /= 2;
	}

	if(cacheindex == 0) gave[blockIdx.x] = buffave[0];
}

//calculate average of voxel values, excluding voxel with too large values, based on average value calculated without filter. Also change the voxel value to average value to reduce artifacts, especially in image recon with normalization.
__global__ void calavewithfilter(float *smatrix, float *gave, int *gnumave)
{
	int msize = d_imageindex[3];
	int cacheindex = threadIdx.x, tid = threadIdx.x + blockIdx.x * blockDim.x;
   	__shared__ float buffave[reducsize];
	__shared__ int buffnumave[reducsize];
    float avepre = aveunf[0];	//average voxel value before filtering

	float buff = 0.0f;
	int buffnum = 0;
	while(tid < msize)
	{
		//if voxelvalue is less than UpholdVox * previous average value, then count it towards new average calculation. Otherwise, change the voxel value to average value to reduce artifacts, especially in image recon with normalization.
		if(smatrix[tid] < UpholdVox * avepre){
		buff += smatrix[tid];
		buffnum += 1;
		}
		else smatrix[tid]  = avepre;

		tid += blockDim.x * gridDim.x ;
	}
	buffave[cacheindex] = buff;
	buffnumave[cacheindex] = buffnum;
	__syncthreads();

	int i = blockDim.x / 2;
	while( i != 0)
	{
		if(cacheindex < i)  {buffave[cacheindex] += buffave[cacheindex + i]; buffnumave[cacheindex] += buffnumave[cacheindex + i];}
		__syncthreads();
		i /= 2;
	}

	if(cacheindex == 0) {gave[blockIdx.x] = buffave[0]; gnumave[blockIdx.x] = buffnumave[0];}
}


//Gaussian blur to image
__global__ void gpublur(float *smatrix, float *bmatrix, float *allweight)
{
	int nx = d_imageindex[0], ny = d_imageindex[1], nz = d_imageindex[2];
    int x = threadIdx.x, y = threadIdx.y, z = blockIdx.x;
    int i; //image index in 1D
    float sumweight, sumval, a = d_info[0];
	int li,hi,lj,hj,lk,hk;
    float	wi;

    while(z < nz)
    {
        y = threadIdx.y;
        while(y < ny)
        {
            x = threadIdx.x;
            while(x < nx)
            {
                i = x + y * nx + z * nx * ny;
				li = max(0, x - d_indr[0]);
				lj = max(0, y - d_indr[1]);
				lk = max(0, z - d_indr[2]);
				hi = min(nx - 1, x + d_indr[0]);
				hj = min(ny - 1, y + d_indr[1]);
				hk = min(nz - 1, z + d_indr[2]);
		        sumweight = 0.0f;
		        sumval = 0.0f;
		
				for(int ii= li; ii<= hi; ii++)
				{
					for(int jj = lj; jj<= hj; jj++)
					{
						for(int kk = lk; kk<= hk; kk++)
						{
							if((powf((ii-x)/d_rads[0],2) + powf((jj-y)/d_rads[1],2) + powf((kk-z)/d_rads[2],2)) * powf(a,2) < 1.0) 
							{
								wi = expf(-(powf(ii-x,2)/(2.0f * powf(d_bsgm[0],2)) + powf(jj-y,2)/(2.0f * powf(d_bsgm[1],2)) + powf(kk-z,2)/(2.0f * powf(d_bsgm[2],2))) * powf(a,2));
								sumval += smatrix[ii + jj * nx + kk * nx * ny] * wi;
								sumweight += wi;
							}
		
						}
					}
				}
				allweight[i] = sumweight;
				bmatrix[i] = sumval / sumweight;
                x += blockDim.x;
            }
            y += blockDim.y;
        }
        z += gridDim.x;
    }


}

// For calculating loglikelihood function value
__global__ void calLogLike(float *xlinevalue, double *gloglike, const int lorindex)
{
	//int msize = d_imageindex[3];
	int cacheindex = threadIdx.x, tid = threadIdx.x + blockIdx.x * blockDim.x;
   	__shared__ double buffave[reducsize];

	double buff = 0.0;
	while(tid < lorindex)
	{
		if (xlinevalue[tid] > ThreshLineValue ) buff += logf(xlinevalue[tid]);
		tid += blockDim.x * gridDim.x ;
	}
	buffave[cacheindex] = buff;
	__syncthreads();

	int i = blockDim.x / 2;
	while( i != 0)
	{
		if(cacheindex < i)  buffave[cacheindex] += buffave[cacheindex + i];
		__syncthreads();
		i /= 2;
	}

	if(cacheindex == 0) gloglike[blockIdx.x] = buffave[0];
}

// For calculating loglikelihood function value 
__global__ void calLogLikeS(float *smatrix, float *normimage, double *gloglike, const int msize, const int norma)
{
	//int msize = d_imageindex[3];
	int cacheindex = threadIdx.x, tid = threadIdx.x + blockIdx.x * blockDim.x;
	float maxnorm = d_norm[0], ThreshNorm = d_norm[1];
   	__shared__ double buffave[reducsize];

	double buff = 0.0;
	while(tid < msize)
	{
		if(norma == 0) buff -= smatrix[tid];
		else {
			float tempSen = normimage[tid] / maxnorm;
			if(tempSen < ThreshNorm) tempSen = ThreshNorm;
			buff -= tempSen * smatrix[tid];
		}
		tid += blockDim.x * gridDim.x ;
	}
	buffave[cacheindex] = buff;
	__syncthreads();

	int i = blockDim.x / 2;
	while( i != 0)
	{
		if(cacheindex < i)  buffave[cacheindex] += buffave[cacheindex + i];
		__syncthreads();
		i /= 2;
	}

	if(cacheindex == 0) gloglike[blockIdx.x] = buffave[0];
}

// For calculating loglikelihood function value 
__global__ void calLogR(float *smatrix, float *poimage, double *gloglike, const int msize)
{
	//int msize = d_imageindex[3];
	int cacheindex = threadIdx.x, tid = threadIdx.x + blockIdx.x * blockDim.x;
   	__shared__ double buffave[reducsize];

	double buff = 0.0;
	while(tid < msize)
	{
		buff -= powf(smatrix[tid]/aves[0] - poimage[tid]/avep[0],2);
		tid += blockDim.x * gridDim.x ;
	}
	buffave[cacheindex] = buff;
	__syncthreads();

	int i = blockDim.x / 2;
	while( i != 0)
	{
		if(cacheindex < i)  buffave[cacheindex] += buffave[cacheindex + i];
		__syncthreads();
		i /= 2;
	}

	if(cacheindex == 0) gloglike[blockIdx.x] = buffave[0];
}
